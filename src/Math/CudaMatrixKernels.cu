#include "hip/hip_runtime.h"
#include "stdio.h"
#include "CudaMatrixKernels.hh"
#include <hip/hip_math_constants.h>
#include <hip/hip_runtime.h>
#include <limits>

#ifdef __CDT_PARSER__
#define __global__
#define __device__
#define __host__
#define __shared__
#endif

#define THREADS_PER_BLOCK 1024

/*****************************************************************************/
/* HELPER FUNCTIONS                                                          */
/*****************************************************************************/

/*
 *
 * atomicAdd for double
 *
 */
 
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif
/*****************************************************************************/


/*
 *
 *  mixed precision axpy
 *
 */


__global__ void __cuda_axpy(int nElements, float alpha, const float *x, double *y){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	y[index] += alpha * x[index];
}

void _cuda_axpy(int nElements, float alpha, const float *x, double *y)
{

    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);
    __cuda_axpy <<< gridSize , THREADS_PER_BLOCK >>> (nElements, alpha, x, y);
}


/*
 *
 *  exp
 *
 */

template<typename T>
__global__ void __cuda_exp(T *data, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] = exp(data[index]);
}

template<typename T>
void _cuda_exp(T *data, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_exp <<< gridSize , THREADS_PER_BLOCK >>> (data, nElements);
}

template __global__ void __cuda_exp<float>(float *, unsigned int);
template __global__ void __cuda_exp<double>(double *, unsigned int);
template void _cuda_exp<float>(float *, unsigned int, unsigned int);
template void _cuda_exp<double>(double *, unsigned int, unsigned int);

/*
 *
 *  signedPow
 *
 */

template<typename T>
__global__ void __cuda_signedPow(T *data, unsigned int nElements, T p){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements) {
        if(data[index] < 0)
            data[index] = -pow(-data[index], p);
        else
            data[index] = pow(data[index], p);
    }
}

template<typename T>
void _cuda_signedPow(T *data, unsigned int nRows, unsigned int nColumns, T p)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_signedPow <<< gridSize , THREADS_PER_BLOCK >>> (data, nElements, p);
}

template __global__ void __cuda_signedPow<float>(float *, unsigned int, float);
template __global__ void __cuda_signedPow<double>(double *, unsigned int, double);
template void _cuda_signedPow<float>(float *, unsigned int, unsigned int, float);
template void _cuda_signedPow<double>(double *, unsigned int, unsigned int, double);

/*
 *
 *  log
 *
 */

template<typename T>
__global__ void __cuda_log(T *data, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] = log(data[index]);
}

template<typename T>
void _cuda_log(T *data, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_log <<< gridSize , THREADS_PER_BLOCK >>> (data, nElements);
}

template __global__ void __cuda_log<float>(float *, unsigned int);
template __global__ void __cuda_log<double>(double *, unsigned int);
template void _cuda_log<float>(float *, unsigned int, unsigned int);
template void _cuda_log<double>(double *, unsigned int, unsigned int);

/*
 *
 *  sin
 *
 */

template<typename T>
__global__ void __cuda_sin(T *data, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] = sin(data[index]);
}

template<typename T>
void _cuda_sin(T *data, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_sin <<< gridSize , THREADS_PER_BLOCK >>> (data, nElements);
}

template __global__ void __cuda_sin<float>(float *, unsigned int);
template __global__ void __cuda_sin<double>(double *, unsigned int);
template void _cuda_sin<float>(float *, unsigned int, unsigned int);
template void _cuda_sin<double>(double *, unsigned int, unsigned int);

/*
 *
 *  cos
 *
 */

template<typename T>
__global__ void __cuda_cos(T *data, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] = cos(data[index]);
}

template<typename T>
void _cuda_cos(T *data, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_cos <<< gridSize , THREADS_PER_BLOCK >>> (data, nElements);
}

template __global__ void __cuda_cos<float>(float *, unsigned int);
template __global__ void __cuda_cos<double>(double *, unsigned int);
template void _cuda_cos<float>(float *, unsigned int, unsigned int);
template void _cuda_cos<double>(double *, unsigned int, unsigned int);

/*
 *
 *  asin
 *
 */

template<typename T>
__global__ void __cuda_asin(T *data, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] = asin(data[index]);
}

template<typename T>
void _cuda_asin(T *data, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_asin <<< gridSize , THREADS_PER_BLOCK >>> (data, nElements);
}

template __global__ void __cuda_asin<float>(float *, unsigned int);
template __global__ void __cuda_asin<double>(double *, unsigned int);
template void _cuda_asin<float>(float *, unsigned int, unsigned int);
template void _cuda_asin<double>(double *, unsigned int, unsigned int);

/*
 *
 *  acos
 *
 */

template<typename T>
__global__ void __cuda_acos(T *data, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] = acos(data[index]);
}

template<typename T>
void _cuda_acos(T *data, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_acos <<< gridSize , THREADS_PER_BLOCK >>> (data, nElements);
}

template __global__ void __cuda_acos<float>(float *, unsigned int);
template __global__ void __cuda_acos<double>(double *, unsigned int);
template void _cuda_acos<float>(float *, unsigned int, unsigned int);
template void _cuda_acos<double>(double *, unsigned int, unsigned int);

/*
 *
 *  abs
 *
 */

template<typename T>
__global__ void __cuda_abs(T *data, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements) {
        if (data[index] < 0)
        	data[index] = -data[index];
    }
}

template<typename T>
void _cuda_abs(T *data, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_abs <<< gridSize , THREADS_PER_BLOCK >>> (data, nElements);
}

template __global__ void __cuda_abs<float>(float *, unsigned int);
template __global__ void __cuda_abs<double>(double *, unsigned int);
template void _cuda_abs<float>(float *, unsigned int, unsigned int);
template void _cuda_abs<double>(double *, unsigned int, unsigned int);

/*
 *
 * tanh
 *
 *
 */

template<typename T>
__global__ void __cuda_tanh(T *data, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] = tanh(data[index]);
}

template<typename T>
void _cuda_tanh(T *data, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_tanh <<< gridSize , THREADS_PER_BLOCK >>> (data, nElements);
}

template __global__ void __cuda_tanh<float>(float *, unsigned int);
template __global__ void __cuda_tanh<double>(double *, unsigned int);
template void _cuda_tanh<float>(float *, unsigned int, unsigned int);
template void _cuda_tanh<double>(double *, unsigned int, unsigned int);

/*
 *
 * sigmoid
 *
 */

template<typename T>
__global__ void __cuda_sigmoid1(T *data, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] = 1.0 / (1.0 + exp(-data[index]));
}

template<typename T>
__global__ void __cuda_sigmoid(T gamma, T *data, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] = 1.0 / (1.0 + exp(-gamma * data[index]));
}

template<typename T>
void _cuda_sigmoid(T gamma, T *data, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);
    if (gamma == 1.0)
	__cuda_sigmoid1 <<< gridSize , THREADS_PER_BLOCK >>> (data, nElements);
    else
	__cuda_sigmoid <<< gridSize , THREADS_PER_BLOCK >>> (gamma, data, nElements);
}

template void _cuda_sigmoid<double>(double gamma, double *data, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_sigmoid<double>(double gamma, double *data, unsigned int nElements);
template __global__ void __cuda_sigmoid1<double>(double *data, unsigned int nElements);
template void _cuda_sigmoid<float>(float gamma, float *data, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_sigmoid<float>(float gamma, float *data, unsigned int nElements);
template __global__ void __cuda_sigmoid1<float>(float *data, unsigned int nElements);

/*
 *
 * triangle
 *
 */

template<typename T>
__global__ void __cuda_triangle(T *data, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements) {
        if ((data[index] < -1.0) || (data[index] > 1.0))
            data[index] = 0.0;
        else if (data[index] < 0.0)
            data[index] = 1.0 + data[index];
        else
            data[index] = 1.0 - data[index];
    }
}

template<typename T>
void _cuda_triangle(T *data, unsigned int nRows, unsigned int nColumns)
{
    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);
	__cuda_triangle <<< gridSize , THREADS_PER_BLOCK >>> (data, nElements);
}

template void _cuda_triangle<double>(double *data, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_triangle<double>(double *data, unsigned int nElements);
template void _cuda_triangle<float>(float *data, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_triangle<float>(float *data, unsigned int nElements);

/*
 *
 * sum
 *
 */
 
template<typename T>
__global__ void __cuda_sum(T *data, unsigned int nRows, unsigned int nColumns, T *result){
    *result = 0;
    for (int i = 0; i < nRows * nColumns; i++){
	*result += data[i];
    }
}

template<typename T>
void _cuda_sum(T *data, unsigned int nRows, unsigned int nColumns, T *result)
{
    // no parallelization, but probably not relevant
    __cuda_sum <<< 1,1>>> (data, nRows, nColumns, result);
}

template __global__ void __cuda_sum<double>(double *data, unsigned int nRows, unsigned int nColumns, double *result);
template void _cuda_sum<double>(double *data, unsigned int nRows, unsigned int nColumns, double *result);
template __global__ void __cuda_sum<float>(float *data, unsigned int nRows, unsigned int nColumns, float *result);
template void _cuda_sum<float>(float *data, unsigned int nRows, unsigned int nColumns, float *result);

/*
 *
 * columnwiseSquaredEuclideanDistance
 *
 */

template<typename T>
__global__ void __cuda_columnwiseSquaredEuclideanDistance(const T *A, unsigned int nRows, unsigned int nColumns, const T *v, T *result){
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nRows * nColumns) {
        T d = A[index] - v[index % nRows];
        d = d*d;
        atomicAdd(&(result[index / nRows]), d);
	}
}

template<typename T>
void _cuda_columnwiseSquaredEuclideanDistance(const T *A, unsigned int nRows, unsigned int nColumns, const T *v, T *result)
{
    int gridSize = (int)ceil( (float) (nRows * nColumns)/THREADS_PER_BLOCK);
	__cuda_columnwiseSquaredEuclideanDistance <<< gridSize , THREADS_PER_BLOCK >>> (A, nRows, nColumns, v, result);
}

template void _cuda_columnwiseSquaredEuclideanDistance<double>(const double *A, unsigned int nRows, unsigned int nColumns, const double *v, double *result);
template __global__ void __cuda_columnwiseSquaredEuclideanDistance<double>(const double *A, unsigned int nRows, unsigned int nColumns, const double *v, double *result);
template void _cuda_columnwiseSquaredEuclideanDistance<float>(const float *A, unsigned int nRows, unsigned int nColumns, const float *v, float *result);
template __global__ void __cuda_columnwiseSquaredEuclideanDistance<float>(const float *A, unsigned int nRows, unsigned int nColumns, const float *v, float *result);


/*
 *
 * clone
 *
 */

template<typename T>
__global__ void __cuda_clone(const T *dataA, T *dataB, unsigned int nRowsB, unsigned int nColumnsB, unsigned int nClones){
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nRowsB * nColumnsB) {
        unsigned int nRowsA = nRowsB / nClones;
        unsigned int rowA = (index % nRowsA);
        unsigned int colA = index / nRowsB;
        dataB[index] = dataA[colA * nRowsA + rowA];
	}
}

template<typename T>
void _cuda_clone(const T *dataA, T *dataB, unsigned int nRowsB, unsigned int nColumnsB, unsigned int nClones)
{
    int nElementsB = nRowsB * nColumnsB;
    int gridSize = (int)ceil( (float) nElementsB/THREADS_PER_BLOCK);
	__cuda_clone <<< gridSize , THREADS_PER_BLOCK >>> (dataA, dataB, nRowsB, nColumnsB, nClones);
}

template void _cuda_clone<double>(const double *dataA, double *dataB, unsigned int nRowsB, unsigned int nColumnsB, unsigned int nClones);
template __global__ void __cuda_clone<double>(const double *dataA, double *dataB, unsigned int nRowsB, unsigned int nColumnsB, unsigned int nClones);
template void _cuda_clone<float>(const float *dataA, float *dataB, unsigned int nRowsB, unsigned int nColumnsB, unsigned int nClones);
template __global__ void __cuda_clone<float>(const float *dataA, float *dataB, unsigned int nRowsB, unsigned int nColumnsB, unsigned int nClones);

/*
 *
 * cloneElementwise
 *
 */

template<typename T>
__global__ void __cuda_cloneElementwise(const T *dataA, T *dataB, unsigned int nElementsB, unsigned int nClones){
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElementsB) {
        unsigned int indexA = index / nClones;
        dataB[index] = dataA[indexA];
	}
}

template<typename T>
void _cuda_cloneElementwise(const T *dataA, T *dataB, unsigned int nRowsB, unsigned int nColumnsB, unsigned int nClones)
{
    int nElementsB = nRowsB * nColumnsB;
    int gridSize = (int)ceil( (float) nElementsB/THREADS_PER_BLOCK);
	__cuda_cloneElementwise <<< gridSize , THREADS_PER_BLOCK >>> (dataA, dataB, nElementsB, nClones);
}

template void _cuda_cloneElementwise<double>(const double *dataA, double *dataB, unsigned int nRowsB, unsigned int nColumnsB, unsigned int nClones);
template __global__ void __cuda_cloneElementwise<double>(const double *dataA, double *dataB, unsigned int nElementsB, unsigned int nClones);
template void _cuda_cloneElementwise<float>(const float *dataA, float *dataB, unsigned int nRowsB, unsigned int nColumnsB, unsigned int nClones);
template __global__ void __cuda_cloneElementwise<float>(const float *dataA, float *dataB, unsigned int nElementsB, unsigned int nClones);

/*
 *
 * addElementsByModuloIndex
 *
 */

template<typename T>
__global__ void __cuda_addElementsByModuloIndex(const T *dataA, T *dataB, unsigned int nRowsA, unsigned int nRowsB, unsigned int nColumns){
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nRowsB * nColumns) {
        unsigned int rowB = index % nRowsB;
        unsigned int column = index / nRowsB;
        for (unsigned int j = 0; j < nRowsA / nRowsB; j++) {
            dataB[index] += dataA[column * nRowsA + (rowB + j * nRowsB)];
        }
	}
}

template<typename T>
void _cuda_addElementsByModuloIndex(const T *dataA, T *dataB, unsigned int nRowsA, unsigned int nRowsB, unsigned int nColumns)
{
    unsigned int nElementsB = nRowsB * nColumns;
    int gridSize = (int)ceil( (float) nElementsB/THREADS_PER_BLOCK);
	__cuda_addElementsByModuloIndex <<< gridSize , THREADS_PER_BLOCK >>> (dataA, dataB, nRowsA, nRowsB, nColumns);
}

template void _cuda_addElementsByModuloIndex<double>(const double *dataA, double *dataB, unsigned int nRowsA, unsigned int nRowsB, unsigned int nColumns);
template __global__ void __cuda_addElementsByModuloIndex<double>(const double *dataA, double *dataB, unsigned int nRowsA, unsigned int nRowsB, unsigned int nColumns);
template void _cuda_addElementsByModuloIndex<float>(const float *dataA, float *dataB, unsigned int nRowsA, unsigned int nRowsB, unsigned int nColumns);
template __global__ void __cuda_addElementsByModuloIndex<float>(const float *dataA, float *dataB, unsigned int nRowsA, unsigned int nRowsB, unsigned int nColumns);


/*
 *
 * chiSquareFeatureMap
 *
 */

template<typename T>
__global__ void __cuda_chiSquareFeatureMap(const T *dataA, T *dataB, unsigned int nElementsB, unsigned int n, T samplingDistance, T min){
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElementsB) {
        unsigned int j = index % (2*n + 1);
        unsigned int baseIndex = index / (2*n + 1);
        T x = (dataA[baseIndex] > min ? dataA[baseIndex] : min);
        if (j == 0) {
            dataB[index] = sqrt(samplingDistance * x);
        }
        else if (j % 2 == 1) {
            T kappa = 1.0 / cosh(HIP_PI * (j+1)/2 * samplingDistance);
            dataB[index] = sqrt(2 * kappa * samplingDistance * x) * cos((j+1)/2 * samplingDistance * log(x));
        }
        else {
            T kappa = 1.0 / cosh(HIP_PI * j/2 * samplingDistance);
            dataB[index] = sqrt(2 * kappa * samplingDistance * x) * sin(j/2 * samplingDistance * log(x));
        }
	}
}

template<typename T>
void _cuda_chiSquareFeatureMap(const T *dataA, T *dataB, unsigned int nElementsB, unsigned int n, T samplingDistance, T min)
{
    int gridSize = (int)ceil( (float) nElementsB/THREADS_PER_BLOCK);
	__cuda_chiSquareFeatureMap <<< gridSize , THREADS_PER_BLOCK >>> (dataA, dataB, nElementsB, n, samplingDistance, min);
}

template void _cuda_chiSquareFeatureMap<double>(const double *dataA, double *dataB, unsigned int nElementsB, unsigned int n, double samplingDistance, double min);
template __global__ void __cuda_chiSquareFeatureMap<double>(const double *dataA, double *dataB, unsigned int nElementsB, unsigned int n, double samplingDistance, double min);
template void _cuda_chiSquareFeatureMap<float>(const float *dataA, float *dataB, unsigned int nElementsB, unsigned int n, float samplingDistance, float min);
template __global__ void __cuda_chiSquareFeatureMap<float>(const float *dataA, float *dataB, unsigned int nElementsB, unsigned int n, float samplingDistance, float min);


/*
 *
 * histogramIntersectionFeatureMap
 *
 */

template<typename T>
__global__ void __cuda_histogramIntersectionFeatureMap(const T *dataA, T *dataB, unsigned int nElementsB, unsigned int n, T samplingDistance, T min){
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElementsB) {
        unsigned int j = index % (2*n + 1);
        unsigned int baseIndex = index / (2*n + 1);
        T x = (dataA[baseIndex] > min ? dataA[baseIndex] : min);
        if (j == 0) {
            dataB[index] = sqrt(2 / HIP_PI * samplingDistance * x);
        }
        else if (j % 2 == 1) {
            T kappa = 2.0 / (HIP_PI * (1 + 4 * (j+1)/2 * samplingDistance * (j+1)/2 * samplingDistance));
            dataB[index] = sqrt(2 * kappa * samplingDistance * x) * cos((j+1)/2 * samplingDistance * log(x));
        }
        else {
            T kappa = 2.0 / (HIP_PI * (1 + 4 * j/2 * samplingDistance * j/2 * samplingDistance));
            dataB[index] = sqrt(2 * kappa * samplingDistance * x) * sin(j/2 * samplingDistance * log(x));
        }
	}
}

template<typename T>
void _cuda_histogramIntersectionFeatureMap(const T *dataA, T *dataB, unsigned int nElementsB, unsigned int n, T samplingDistance, T min)
{
    int gridSize = (int)ceil( (float) nElementsB/THREADS_PER_BLOCK);
	__cuda_histogramIntersectionFeatureMap <<< gridSize , THREADS_PER_BLOCK >>> (dataA, dataB, nElementsB, n, samplingDistance, min);
}

template void _cuda_histogramIntersectionFeatureMap<double>(const double *dataA, double *dataB, unsigned int nElementsB, unsigned int n, double samplingDistance, double min);
template __global__ void __cuda_histogramIntersectionFeatureMap<double>(const double *dataA, double *dataB, unsigned int nElementsB, unsigned int n, double samplingDistance, double min);
template void _cuda_histogramIntersectionFeatureMap<float>(const float *dataA, float *dataB, unsigned int nElementsB, unsigned int n, float samplingDistance, float min);
template __global__ void __cuda_histogramIntersectionFeatureMap<float>(const float *dataA, float *dataB, unsigned int nElementsB, unsigned int n, float samplingDistance, float min);


/*
 *
 * elementwiseMultiplicationWithChiSquareFeatureMapDerivative
 *
 */

template<typename T>
__global__ void __cuda_elementwiseMultiplicationWithApproximateFeatureMapDerivative(const T *dataA, T *dataB, unsigned int nElements, unsigned int n, T samplingDistance, T kappa0){
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements) {
        unsigned int j = index % (2 * n + 1);
        if (j == 0) {
            dataB[index] *= dataA[index];
        }
        else if (j % 2 == 1) {
            dataB[index] *= dataA[index] - (j+1) * samplingDistance * dataA[index + 1];
        }
        else {
            dataB[index] *= dataA[index] + j * samplingDistance * dataA[index - 1];
        }
        dataB[index] *= samplingDistance * kappa0 / (2.0 * dataA[index - j] * dataA[index - j]);
	}
}

template<typename T>
void _cuda_elementwiseMultiplicationWithApproximateFeatureMapDerivative(const T *dataA, T *dataB, unsigned int nElements, unsigned int n, T samplingDistance, T kappa0)
{
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);
	__cuda_elementwiseMultiplicationWithApproximateFeatureMapDerivative <<< gridSize , THREADS_PER_BLOCK >>> (dataA, dataB, nElements, n, samplingDistance, kappa0);
}

template void _cuda_elementwiseMultiplicationWithApproximateFeatureMapDerivative<double>(const double *dataA, double *dataB, unsigned int nElements, unsigned int n, double samplingDistance, double kappa0);
template __global__ void __cuda_elementwiseMultiplicationWithApproximateFeatureMapDerivative<double>(const double *dataA, double *dataB, unsigned int nElements, unsigned int n, double samplingDistance, double kappa0);
template void _cuda_elementwiseMultiplicationWithApproximateFeatureMapDerivative<float>(const float *dataA, float *dataB, unsigned int nElements, unsigned int n, float samplingDistance, float kappa0);
template __global__ void __cuda_elementwiseMultiplicationWithApproximateFeatureMapDerivative<float>(const float *dataA, float *dataB, unsigned int nElements, unsigned int n, float samplingDistance, float kappa0);


/*
 *
 * addSummedRows
 *
 */
template<typename T>
__global__ void __cuda_addSummedRows(T *vectorDevPtr, const T *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const T scale){
    unsigned  int columnIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (columnIndex < nColumns){
	float result = 0.0;
	for (unsigned int i = 0; i < nRows; i++){
	    // result += matrix(i,columnIndex)
	    result += matrixDevPtr[columnIndex * nRows + i];
	}
	vectorDevPtr[columnIndex] += scale * result;
    }
}

template<typename T>
void _cuda_addSummedRows(T *vectorDevPtr, const T *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const T scale){

    // parallelize over columns
    int gridSize = (int)ceil( (float) nColumns/THREADS_PER_BLOCK);

    __cuda_addSummedRows <<< gridSize , THREADS_PER_BLOCK >>> (vectorDevPtr, matrixDevPtr, nRows, nColumns, scale);
}

template __global__ void __cuda_addSummedRows(double *vectorDevPtr, const double *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const double scale);
template void _cuda_addSummedRows(double *vectorDevPtr, const double *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const double scale);
template __global__ void __cuda_addSummedRows(float *vectorDevPtr, const float *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const float scale);
template void _cuda_addSummedRows(float *vectorDevPtr, const float *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const float scale);

/*
 * slightly faster version using tmp array
 *
 */
template<typename T>
__global__ void __cuda_summedRowsTmp(const T *matrixDevPtr, unsigned int nRows, unsigned int nColumns,
	T *tmpDevPtr, unsigned int tmpRows){
    unsigned int columnIndex = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int columnPart = blockIdx.y;
    if (columnIndex < nColumns){
	unsigned int nRowsDiv = nRows / tmpRows;
	unsigned int startRow =  columnPart * nRowsDiv;
	if (startRow < nRows){
	    unsigned int endRow = columnPart == tmpRows - 1 ? nRows : (columnPart + 1) * nRowsDiv;
	    T result = 0.0;
	    for (unsigned int i = startRow; i < endRow; i++){
		// result += matrix(i, columnIndex)
		result += matrixDevPtr[columnIndex * nRows + i];
	    }
	    tmpDevPtr[columnIndex*tmpRows + columnPart] = result;
	}
    }
}

template<typename T>
void _cuda_addSummedRows(T *vectorDevPtr, const T *matrixDevPtr, unsigned int nRows, unsigned int nColumns,
	T *tmpDevPtr, unsigned int tmpRows, const T scale){
    int gridDimx = (int)ceil( (float) nColumns / THREADS_PER_BLOCK);
    int gridDimy = tmpRows;
    dim3 gridSize(gridDimx,gridDimy);
    __cuda_summedRowsTmp <<< gridSize , THREADS_PER_BLOCK >>> (matrixDevPtr, nRows, nColumns, tmpDevPtr, tmpRows);

    _cuda_addSummedRows<T>(vectorDevPtr, tmpDevPtr, tmpRows, nColumns, scale);
}

template __global__ void __cuda_summedRowsTmp<double>(const double *matrixDevPtr, unsigned int nRows, unsigned int nColumns,
	double *tmpDevPtr, unsigned int tmpRows);
template void _cuda_addSummedRows<double>(double *vectorDevPtr, const double *matrixDevPtr, unsigned int nRows, unsigned int nColumns,
	double *tmpDevPtr, unsigned int tmpRows, const double scale);
template __global__ void __cuda_summedRowsTmp<float>(const float *matrixDevPtr, unsigned int nRows, unsigned int nColumns,
	float *tmpDevPtr, unsigned int tmpRows);
template void _cuda_addSummedRows<float>(float *vectorDevPtr, const float *matrixDevPtr, unsigned int nRows, unsigned int nColumns,
	float *tmpDevPtr, unsigned int tmpRows, const float scale);
	
/*
 *
 * addSummedColumns
 *
 */

template<typename T>
__global__ void __cuda_addSummedColumns(T *vectorDevPtr, const T *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const T scale){
    unsigned  int rowIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (rowIndex < nRows){
	T result = 0.0;
	for (unsigned int i = 0; i < nColumns; i++){
	    // result += matrix(rowIndex,i)
	    result += matrixDevPtr[i * nRows + rowIndex];
	}
	vectorDevPtr[rowIndex] += scale * result;
    }
}

template<typename T>
void _cuda_addSummedColumns(T *vectorDevPtr, const T *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const T scale){
    // parallelize over rows
    int gridSize = (int)ceil( (float) nRows/THREADS_PER_BLOCK);

    __cuda_addSummedColumns <<< gridSize , THREADS_PER_BLOCK >>> (vectorDevPtr, matrixDevPtr, nRows, nColumns, scale);
}

template __global__ void __cuda_addSummedColumns<double>(double *vectorDevPtr, const double *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const double scale);
template     void _cuda_addSummedColumns<double>(double *vectorDevPtr, const double *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const double scale);
template __global__ void __cuda_addSummedColumns<float>(float *vectorDevPtr, const float *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const float scale);
template     void _cuda_addSummedColumns<float>(float *vectorDevPtr, const float *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const float scale);

/*
 * addSummedColumnsChannelWise
 *
 *
 */
template<typename T>
__global__ void __cuda_addSummedColumnsChannelWise(T *vector, const T* matrix, const unsigned int channels, const unsigned int nRows, const unsigned int nColumns, const T scale)
{
	unsigned int channelIndex = threadIdx.x + blockIdx.x * blockDim.x;
	if(channelIndex < channels) {
		unsigned int channelSize = nRows / channels;
		for(unsigned int i=0; i < channelSize; i++) {
			for(unsigned int j=0; j < nColumns; j++) {
				vector[channelIndex] += scale * matrix[j * nRows + channelIndex * channelSize + i];
			}
		}
	}
}
template<typename T>
void _cuda_addSummedColumnsChannelWise(T *vector, const T* matrix, const unsigned int channels, const unsigned int nRows, const unsigned int nColumns, const T scale)
{
	int gridSize = (int)ceil( (float) channels/THREADS_PER_BLOCK);
	__cuda_addSummedColumnsChannelWise<<<gridSize, THREADS_PER_BLOCK>>>(vector, matrix, channels, nRows, nColumns, scale);
}
template __global__ void __cuda_addSummedColumnsChannelWise(double *vector, const double* matrix, const unsigned int channels, const unsigned int nRows, const unsigned int nColumns, const double scale);
template __global__ void __cuda_addSummedColumnsChannelWise(float *vector, const float* matrix, const unsigned int channels, const unsigned int nRows, const unsigned int nColumns, const float scale);
template void _cuda_addSummedColumnsChannelWise(double *vector, const double* matrix, const unsigned int channels, const unsigned int nRows, const unsigned int nColumns, const double scale);
template void _cuda_addSummedColumnsChannelWise(float *vector, const float* matrix, const unsigned int channels, const unsigned int nRows, const unsigned int nColumns, const float scale);
/*
 *
 * addSquaredSummedColumns
 *
 */

template<typename T>
__global__ void __cuda_addSquaredSummedColumns(T *vectorDevPtr, const T *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const T scale){
    unsigned  int rowIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (rowIndex < nRows){
	T result = 0.0;
	for (unsigned int i = 0; i < nColumns; i++){
	    result += matrixDevPtr[i * nRows + rowIndex] * matrixDevPtr[i * nRows + rowIndex];
	}
	vectorDevPtr[rowIndex] += scale * result;
    }
}

template<typename T>
void _cuda_addSquaredSummedColumns(T *vectorDevPtr, const T *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const T scale){

    // parallelize over rows
    int gridSize = (int)ceil( (float) nRows/THREADS_PER_BLOCK);

    __cuda_addSquaredSummedColumns <<< gridSize , THREADS_PER_BLOCK >>> (vectorDevPtr, matrixDevPtr, nRows, nColumns, scale);
}

template __global__ void __cuda_addSquaredSummedColumns(double *vectorDevPtr, const double *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const double scale);
template void _cuda_addSquaredSummedColumns(double *vectorDevPtr, const double *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const double scale);
template __global__ void __cuda_addSquaredSummedColumns(float *vectorDevPtr, const float *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const float scale);
template void _cuda_addSquaredSummedColumns(float *vectorDevPtr, const float *matrixDevPtr, unsigned int nRows, unsigned int nColumns, const float scale);

/*
 *
 * addSummedNeighborsInARow
 *
 */

template<typename T>
__global__ void __cuda_addSummedNeighborsInARow(T* dataA, const T* dataB, unsigned int elementsA, unsigned int nNeighbors){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < elementsA){
        for (unsigned int n = 0; n < nNeighbors; n++){
            dataA[index] += dataB[index * nNeighbors + n];
        }
    }
}

template<typename T>
void _cuda_addSummedNeighborsInARow(T* dataA, const T* dataB, unsigned int rowsA, unsigned int columnsA, unsigned int nNeighbors){

    // parallelize over rows
    int gridSize = (int)ceil( (float) rowsA*columnsA/THREADS_PER_BLOCK);

    __cuda_addSummedNeighborsInARow <<< gridSize , THREADS_PER_BLOCK >>> (dataA, dataB, rowsA * columnsA, nNeighbors);
}

template __global__ void __cuda_addSummedNeighborsInARow(double* dataA, const double* dataB, unsigned int elementsA, unsigned int nNeighbors);
template void _cuda_addSummedNeighborsInARow(double* dataA, const double* dataB, unsigned int rowsA, unsigned int columnsA, unsigned int nNeighbors);
template __global__ void __cuda_addSummedNeighborsInARow(float* dataA, const float* dataB, unsigned int elementsA, unsigned int nNeighbors);
template void _cuda_addSummedNeighborsInARow(float* dataA, const float* dataB, unsigned int rowsA, unsigned int columnsA, unsigned int nNeighbors);


/*
 *
 * addWeighted
 *
 */

template<typename T>
__global__ void __cuda_addWeighted(T *data, const T *X, const T* weights, unsigned int nRows, unsigned int nColumns){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nRows * nColumns) {
    	unsigned int col = index / nRows;
		data[index] += X[index] * weights[col];
	}
}

template<typename T>
void _cuda_addWeighted(T *data, const T *X, const T* weights, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_addWeighted <<< gridSize , THREADS_PER_BLOCK >>> (data, X, weights, nRows, nColumns);
}

template __global__ void __cuda_addWeighted<double>(double *data, const double *X, const double* weights, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_addWeighted<float>(float *data, const float *X, const float* weights, unsigned int nRows, unsigned int nColumns);
template void _cuda_addWeighted<double>(double *data, const double *X, const double* weights, unsigned int nRows, unsigned int nColumns);
template void _cuda_addWeighted<float>(float *data, const float *X, const float* weights, unsigned int nRows, unsigned int nColumns);


/*
 *
 * elementwise multiplication
 *
 */

template<typename T>
__global__ void __cuda_elementwiseMultiplication(T *data, T *datab, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] = data[index] * datab[index];
}

template<typename T>
void _cuda_elementwiseMultiplication(T *data, T *datab, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_elementwiseMultiplication <<< gridSize , THREADS_PER_BLOCK >>> (data, datab, nElements);
}

template __global__ void __cuda_elementwiseMultiplication<double>(double *data, double *datab, unsigned int nElements);
template __global__ void __cuda_elementwiseMultiplication<float>(float *data, float *datab, unsigned int nElements);
template void _cuda_elementwiseMultiplication<double>(double *data, double *datab, unsigned int nRows, unsigned int nColumns);
template void _cuda_elementwiseMultiplication<float>(float *data, float *datab, unsigned int nRows, unsigned int nColumns);

/*
 *
 * elementwise division
 *
 */

template<typename T>
__global__ void __cuda_elementwiseDivision(T *data, T *datab, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] = data[index] / datab[index];
}

template<typename T>
void _cuda_elementwiseDivision(T *data, T *datab, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_elementwiseDivision <<< gridSize , THREADS_PER_BLOCK >>> (data, datab, nElements);
}

template __global__ void __cuda_elementwiseDivision<double>(double *data, double *datab, unsigned int nElements);
template __global__ void __cuda_elementwiseDivision<float>(float *data, float *datab, unsigned int nElements);
template void _cuda_elementwiseDivision<double>(double *data, double *datab, unsigned int nRows, unsigned int nColumns);
template void _cuda_elementwiseDivision<float>(float *data, float *datab, unsigned int nRows, unsigned int nColumns);


/*
 *
 * rprop Weight Update
 *
 */

template<typename T>
__global__ void __cuda_rpropUpdate(T *currentValues, T *newGradients, T *oldGradients, T *updateValues, T increasingFactor, T decreasingFactor, T maxUpdateValue, T minUpdateValue, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements) {
		T change = oldGradients[index] *  newGradients[index];
		if (change > 0) {
			updateValues[index] = updateValues[index] * increasingFactor;
			if (updateValues[index] > maxUpdateValue)
				updateValues[index] = maxUpdateValue;
		} else if (change < 0) {
			updateValues[index] = updateValues[index] * decreasingFactor;
			if (updateValues[index] < minUpdateValue)
				updateValues[index] = minUpdateValue;
		}
		if (newGradients[index] > 0)
			currentValues[index] = currentValues[index] - updateValues[index];
		else if (newGradients[index] < 0)
			currentValues[index] = currentValues[index] + updateValues[index];
		oldGradients[index] = newGradients[index];
	}
}

template<typename T>
void _cuda_rpropUpdate(T *currentValues, T *newGradients, T *oldGradients, T *updateValues, T increasingFactor, T decreasingFactor, T maxUpdateValue, T minUpdateValue, unsigned int nRows, unsigned int nColumns)
{
    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_rpropUpdate <<< gridSize , THREADS_PER_BLOCK >>> (currentValues, newGradients, oldGradients, updateValues, increasingFactor, decreasingFactor, maxUpdateValue, minUpdateValue, nElements);
}

template __global__ void __cuda_rpropUpdate<double>(double *currentValues, double *newGradients, double *oldGradients, double *updateValues, double increasingFactor, double decreasingFactor, double maxUpdateValue, double minUpdateValue, unsigned int nElements);
template __global__ void __cuda_rpropUpdate<float>(float *currentValues, float *newGradients, float *oldGradients, float *updateValues, float increasingFactor, float decreasingFactor, float maxUpdateValue, float minUpdateValue, unsigned int nElements);
template void _cuda_rpropUpdate<double>(double *currentValues, double *newGradients, double *oldGradients, double *updateValues, double increasingFactor, double decreasingFactor, double maxUpdateValue, double minUpdateValue, unsigned int nRows, unsigned int nColumns);
template void _cuda_rpropUpdate<float>(float *currentValues, float *newGradients, float *oldGradients, float *updateValues, float increasingFactor, float decreasingFactor, float maxUpdateValue, float minUpdateValue, unsigned int nRows, unsigned int nColumns);


/*
 *
 * add constant elementwise
 *
 */
template<typename T>
__global__ void __cuda_addConstantElementwise(T constant, T *data, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] = data[index] + constant;
}

template<typename T>
void _cuda_addConstantElementwise(T constant, T *data, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (T) nElements/THREADS_PER_BLOCK);

    __cuda_addConstantElementwise <<< gridSize , THREADS_PER_BLOCK >>> (constant, data, nElements);
}

template __global__ void __cuda_addConstantElementwise<double>(double constant, double *data, unsigned int nElements);
template void _cuda_addConstantElementwise<double>(double constant, double *data, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_addConstantElementwise<float>(float constant, float *data, unsigned int nElements);
template void _cuda_addConstantElementwise<float>(float constant, float *data, unsigned int nRows, unsigned int nColumns);


/*
 *
 * getMaxOfColumns
 *
 */
template<typename T>
__global__ void __cuda_getMaxOfColumns(T *vectorDevPtr, const T *matrixDevPtr, unsigned int nRows, unsigned int nColumns){
    unsigned  int columnIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (columnIndex < nColumns){
	T result = matrixDevPtr[columnIndex * nRows];
	for (unsigned int i = 1; i < nRows; i++){
	    T val = matrixDevPtr[columnIndex * nRows + i];
	    result = fmax(result, val);
	}
	vectorDevPtr[columnIndex] = result;
    }
}

template<typename T>
void _cuda_getMaxOfColumns(T *vectorDevPtr, const T *matrixDevPtr, unsigned int nRows, unsigned int nColumns){
    // parallelize over columns
    int gridSize = (int)ceil( (float) nColumns/THREADS_PER_BLOCK);

    __cuda_getMaxOfColumns <<< gridSize , THREADS_PER_BLOCK >>> (vectorDevPtr, matrixDevPtr, nRows, nColumns);
}

template __global__ void __cuda_getMaxOfColumns(double *vectorDevPtr, const double *matrixDevPtr, unsigned int nRows, unsigned int nColumns);
template void _cuda_getMaxOfColumns(double *vectorDevPtr, const double *matrixDevPtr, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_getMaxOfColumns(float *vectorDevPtr, const float *matrixDevPtr, unsigned int nRows, unsigned int nColumns);
template void _cuda_getMaxOfColumns(float *vectorDevPtr, const float *matrixDevPtr, unsigned int nRows, unsigned int nColumns);

/*
 * slightly faster version using tmp array
 */

template<typename T>
__global__ void __cuda_getMaxOfColumnsTmp(const T *matrixDevPtr, unsigned int nRows, unsigned int nColumns,
	T *tmpDevPtr, unsigned int tmpRows){
    unsigned int columnIndex = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int columnPart = blockIdx.y;
    if (columnIndex < nColumns){
	unsigned int nRowsDiv = nRows / tmpRows;
	unsigned int startRow =  columnPart * nRowsDiv;
	if (startRow < nRows){
	    unsigned int endRow = columnPart == tmpRows - 1 ? nRows : (columnPart + 1) * nRowsDiv;
	    T result = matrixDevPtr[columnIndex * nRows];
	    for (unsigned int i = startRow; i < endRow; i++){
		// result += matrix(i, columnIndex)
		T val = matrixDevPtr[columnIndex * nRows + i];
		result = fmax(result, val);
	    }
	    tmpDevPtr[columnIndex*tmpRows + columnPart] = result;
	}
    }
}

template<typename T>
void _cuda_getMaxOfColumns(T *vectorDevPtr, const T *matrixDevPtr, unsigned int nRows, unsigned int nColumns,
	T *tmpDevPtr, unsigned int tmpRows){
    int gridDimx = (int)ceil( (float) nColumns / THREADS_PER_BLOCK);
    int gridDimy = tmpRows;
    dim3 gridSize(gridDimx,gridDimy);

    __cuda_getMaxOfColumnsTmp <<< gridSize , THREADS_PER_BLOCK >>> (matrixDevPtr, nRows, nColumns, tmpDevPtr, tmpRows);

    _cuda_getMaxOfColumns<T>(vectorDevPtr, tmpDevPtr, tmpRows, nColumns);
}

template __global__ void __cuda_getMaxOfColumnsTmp(const double *matrixDevPtr, unsigned int nRows, unsigned int nColumns,
	double *tmpDevPtr, unsigned int tmpRows);
template void _cuda_getMaxOfColumns(double *vectorDevPtr, const double *matrixDevPtr, unsigned int nRows, unsigned int nColumns,
	double *tmpDevPtr, unsigned int tmpRows);
template __global__ void __cuda_getMaxOfColumnsTmp(const float *matrixDevPtr, unsigned int nRows, unsigned int nColumns,
	float *tmpDevPtr, unsigned int tmpRows);
template void _cuda_getMaxOfColumns(float *vectorDevPtr, const float *matrixDevPtr, unsigned int nRows, unsigned int nColumns,
	float *tmpDevPtr, unsigned int tmpRows);

/*
 *
 * elementwiseMultiplicationWithSigmoidDerivative
 *
 */

template<typename T>
__global__ void __cuda_elementwiseMultiplicationWithSigmoidDerivative(T *data, T *datab, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] = data[index] * (datab[index] * (1 - datab[index]));
}

template<typename T>
void _cuda_elementwiseMultiplicationWithSigmoidDerivative(T *data, T *datab, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_elementwiseMultiplicationWithSigmoidDerivative <<< gridSize , THREADS_PER_BLOCK >>> (data, datab, nElements);
}

template __global__ void __cuda_elementwiseMultiplicationWithSigmoidDerivative(double *data, double *datab, unsigned int nElements);
template void _cuda_elementwiseMultiplicationWithSigmoidDerivative(double *data, double *datab, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_elementwiseMultiplicationWithSigmoidDerivative(float *data, float *datab, unsigned int nElements);
template void _cuda_elementwiseMultiplicationWithSigmoidDerivative(float *data, float *datab, unsigned int nRows, unsigned int nColumns);

/*
 *
 * elementwiseMultiplicationWithTriangleDerivative
 *
 */

template<typename T>
__global__ void __cuda_elementwiseMultiplicationWithTriangleDerivative(T *data, T *datab, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements) {
        if ((datab[index] < -1.0) || (datab[index] > 1.0) || (datab[index] == 0))
        	data[index] = 0;
        else if (datab[index] > 0.0)
        	data[index] = -data[index];
    }
}

template<typename T>
void _cuda_elementwiseMultiplicationWithTriangleDerivative(T *data, T *datab, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_elementwiseMultiplicationWithTriangleDerivative <<< gridSize , THREADS_PER_BLOCK >>> (data, datab, nElements);
}

template __global__ void __cuda_elementwiseMultiplicationWithTriangleDerivative(double *data, double *datab, unsigned int nElements);
template void _cuda_elementwiseMultiplicationWithTriangleDerivative(double *data, double *datab, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_elementwiseMultiplicationWithTriangleDerivative(float *data, float *datab, unsigned int nElements);
template void _cuda_elementwiseMultiplicationWithTriangleDerivative(float *data, float *datab, unsigned int nRows, unsigned int nColumns);

/*
 *
 * elementwiseMultiplicationWithTanhDerivative
 *
 */

template<typename T>
__global__ void __cuda_elementwiseMultiplicationWithTanhDerivative(T *data, T *datab, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] = data[index] * (1 - pow(datab[index],2));
}

template<typename T>
void _cuda_elementwiseMultiplicationWithTanhDerivative(T *data, T *datab, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_elementwiseMultiplicationWithTanhDerivative <<< gridSize , THREADS_PER_BLOCK >>> (data, datab, nElements);
}

template __global__ void __cuda_elementwiseMultiplicationWithTanhDerivative(double *data, double *datab, unsigned int nElements);
template void _cuda_elementwiseMultiplicationWithTanhDerivative(double *data, double *datab, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_elementwiseMultiplicationWithTanhDerivative(float *data, float *datab, unsigned int nElements);
template void _cuda_elementwiseMultiplicationWithTanhDerivative(float *data, float *datab, unsigned int nRows, unsigned int nColumns);

/*
 *
 * multiplicationWithSoftmaxDerivative
 *
 */

template<typename T>
__global__ void __cuda_multiplicationWithSoftmaxDerivative(T *data, T *datab, T *datac, unsigned int nElements, unsigned int nRows){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] = datab[index] * (data[index] - datac[index/nRows]);
}

template<typename T>
void _cuda_multiplicationWithSoftmaxDerivative(T *data, T *datab, T *datac, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_multiplicationWithSoftmaxDerivative <<< gridSize , THREADS_PER_BLOCK >>> (data, datab, datac, nElements, nRows);
}

template __global__ void __cuda_multiplicationWithSoftmaxDerivative(double *data, double *datab, double *datac, unsigned int nElements, unsigned int nRows);
template void _cuda_multiplicationWithSoftmaxDerivative(double *data, double *datab, double *datac, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_multiplicationWithSoftmaxDerivative(float *data, float *datab, float *datac, unsigned int nElements, unsigned int nRows);
template void _cuda_multiplicationWithSoftmaxDerivative(float *data, float *datab, float *datac, unsigned int nRows, unsigned int nColumns);


/*
 * elementwiseMultiplicationWithClippedDerivative
 *
 */

template <typename T>
__global__ void __cuda_elementwiseMultiplicationWithClippedDerivative(T *errOut, T *activations, unsigned int nElements, T thresholdLeft, T thresholdRight){
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements) {
		if ((activations[index] <= thresholdLeft) || (activations[index] >= thresholdRight)) errOut[index] = 0;
	}
}
template <typename T>
void _cuda_elementwiseMultiplicationWithClippedDerivative(T *data, T *datab, unsigned int nRows, unsigned int nColumns, T thresholdLeft, T thresholdRight) {
    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);
    __cuda_elementwiseMultiplicationWithClippedDerivative<T> <<<gridSize, THREADS_PER_BLOCK>>> (data, datab, nElements, thresholdLeft, thresholdRight);
}
template __global__ void __cuda_elementwiseMultiplicationWithClippedDerivative<float>(float*, float*, unsigned int, float, float);
template __global__ void __cuda_elementwiseMultiplicationWithClippedDerivative<double>(double*, double*, unsigned int, double, double);
template void _cuda_elementwiseMultiplicationWithClippedDerivative<float>(float*, float*, unsigned int, unsigned int, float, float);
template void _cuda_elementwiseMultiplicationWithClippedDerivative<double>(double*, double*, unsigned int, unsigned int, double, double);


/*
 * elementwiseMultiplicationWithSignedPowDerivative
 *
 */

template <typename T>
__global__ void __cuda_elementwiseMultiplicationWithSignedPowDerivative(T *errOut, T *activations, unsigned int nElements, T p){
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements) {
        if (activations[index] == 0)
            errOut[index] = 0;
        else if (activations[index] < 0)
            errOut[index] *= p * pow(-activations[index], p - 1);
        else
            errOut[index] *= p * pow(activations[index], p - 1);
    }
}
template <typename T>
void _cuda_elementwiseMultiplicationWithSignedPowDerivative(T *data, T *datab, unsigned int nRows, unsigned int nColumns, T p) {
    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);
    __cuda_elementwiseMultiplicationWithSignedPowDerivative<T> <<<gridSize, THREADS_PER_BLOCK>>> (data, datab, nElements, p);
}
template __global__ void __cuda_elementwiseMultiplicationWithSignedPowDerivative<float>(float*, float*, unsigned int, float);
template __global__ void __cuda_elementwiseMultiplicationWithSignedPowDerivative<double>(double*, double*, unsigned int, double);
template void _cuda_elementwiseMultiplicationWithSignedPowDerivative<float>(float*, float*, unsigned int, unsigned int, float);
template void _cuda_elementwiseMultiplicationWithSignedPowDerivative<double>(double*, double*, unsigned int, unsigned int, double);


/*
 * elementwiseMultiplicationWithLogDerivative
 *
 */

template <typename T>
__global__ void __cuda_elementwiseMultiplicationWithLogDerivative(T *errOut, T *activations, unsigned int nElements){
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        errOut[index] *= exp(-activations[index]);
}
template <typename T>
void _cuda_elementwiseMultiplicationWithLogDerivative(T *data, T *datab, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);
    __cuda_elementwiseMultiplicationWithLogDerivative<T> <<<gridSize, THREADS_PER_BLOCK>>> (data, datab, nElements);
}
template __global__ void __cuda_elementwiseMultiplicationWithLogDerivative<float>(float*, float*, unsigned int);
template __global__ void __cuda_elementwiseMultiplicationWithLogDerivative<double>(double*, double*, unsigned int);
template void _cuda_elementwiseMultiplicationWithLogDerivative<float>(float*, float*, unsigned int, unsigned int);
template void _cuda_elementwiseMultiplicationWithLogDerivative<double>(double*, double*, unsigned int, unsigned int);


/*
 *
 * multiplicationWithL2NormalizationDerivative
 *
 */

template<typename T>
__global__ void __cuda_multiplicationWithL2NormalizationDerivative(T *data, T *datab, T *datac, T *datad, unsigned int nElements, unsigned int nRows){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
    data[index] = (data[index] - datab[index] * datac[index/nRows]) / datad[index/nRows];
}

template<typename T>
void _cuda_multiplicationWithL2NormalizationDerivative(T *data, T *datab, T *datac, T *datad, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_multiplicationWithL2NormalizationDerivative <<< gridSize , THREADS_PER_BLOCK >>> (data, datab, datac, datad, nElements, nRows);
}

template __global__ void __cuda_multiplicationWithL2NormalizationDerivative(double *data, double *datab, double *datac, double *datad, unsigned int nElements, unsigned int nRows);
template void _cuda_multiplicationWithL2NormalizationDerivative(double *data, double *datab, double *datac, double *datad, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_multiplicationWithL2NormalizationDerivative(float *data, float *datab, float *datac, float *datad, unsigned int nElements, unsigned int nRows);
template void _cuda_multiplicationWithL2NormalizationDerivative(float *data, float *datab, float *datac, float *datad, unsigned int nRows, unsigned int nColumns);


/*
 *
 * addToAllColumns
 *
 */


template<typename T>
__global__ void __cuda_addToAllColumns(T *data, T *datab, unsigned int nElements, unsigned int nRows, T alpha){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] += alpha * datab[index%nRows];
}

template<typename T>
void _cuda_addToAllColumns(T *data, T *datab, unsigned int nRows, unsigned int nColumns, T alpha)
{
    // TODO implement kernel without % operator (slow on GPU)
    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_addToAllColumns <<< gridSize , THREADS_PER_BLOCK >>> (data, datab, nElements, nRows, alpha);
}

template __global__ void __cuda_addToAllColumns<double>(double *data, double *datab, unsigned int nElements, unsigned int nRows, double alpha);
template void _cuda_addToAllColumns<double>(double *data, double *datab, unsigned int nRows, unsigned int nColumns, double alpha);
template __global__ void __cuda_addToAllColumns<float>(float *data, float *datab, unsigned int nElements, unsigned int nRows, float alpha);
template void _cuda_addToAllColumns<float>(float *data, float *datab, unsigned int nRows, unsigned int nColumns, float alpha);

/*
 * addToAllChannels
 * Adds one element of vector to one channel
 */
template<typename T>
__global__ void __cuda_addToAllChannels(T *mat, T *vec, unsigned int channels, unsigned int nRows, unsigned int nElements, T alpha)
{
	unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < nElements) {
		unsigned int channelSize = nRows / channels;
		mat[index] += alpha * vec[(index%nRows)/channelSize];
	}
}
template<typename T>
void _cuda_addToAllChannels(T *mat, T *vec, unsigned int channels, unsigned int nRows, unsigned int nColumns, T alpha)
{
	unsigned int nElements = nRows * nColumns;
	int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);
	
	__cuda_addToAllChannels<<<gridSize, THREADS_PER_BLOCK>>>(mat, vec, channels, nRows, nElements, alpha);
}
template __global__ void __cuda_addToAllChannels(double *mat, double *vec, unsigned int channels, unsigned int nRows, unsigned int nElements, double alpha);
template __global__ void __cuda_addToAllChannels(float *mat, float *vec, unsigned int channels, unsigned int nRows, unsigned int nElements, float alpha);
template void _cuda_addToAllChannels(double *mat, double *vec, unsigned int channels, unsigned int nRows, unsigned int nColumns, double alpha);
template void _cuda_addToAllChannels(float *mat, float *vec, unsigned int channels, unsigned int nRows, unsigned int nColumns, float alpha);
/*
 *
 * addToAllRows
 *
 */
template<typename T>
__global__ void __cuda_addToAllRows(T *data, T *datab, unsigned int nElements, unsigned int nRows, T alpha){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] += alpha * datab[index/nRows];
}
template<typename T>
void _cuda_addToAllRows(T *data, T *datab, unsigned int nRows, unsigned int nColumns, T alpha)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_addToAllRows <<< gridSize , THREADS_PER_BLOCK >>> (data, datab, nElements, nRows, alpha);
}

template __global__ void __cuda_addToAllRows<double>(double *data, double *datab, unsigned int nElements, unsigned int nRows, double alpha);
template void _cuda_addToAllRows<double>(double *data, double *datab, unsigned int nRows, unsigned int nColumns, double alpha);
template __global__ void __cuda_addToAllRows<float>(float *data, float *datab, unsigned int nElements, unsigned int nRows, float alpha);
template void _cuda_addToAllRows<float>(float *data, float *datab, unsigned int nRows, unsigned int nColumns, float alpha);

/*
 *
 * multiplyColumnsByScalars
 *
 */
template<typename T>
__global__ void __cuda_multiplyColumnsByScalars(const T *vectorDevPtr, T *matrixDevPtr, unsigned int nRows, unsigned int nElements){
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int colIndex = index / nRows;
    if (index < nElements)
	matrixDevPtr[index] = matrixDevPtr[index] * vectorDevPtr[colIndex];
}
template<typename T>
void _cuda_multiplyColumnsByScalars(const T *vectorDevPtr, T *matrixDevPtr, unsigned int nRows, unsigned int nColumns){
    // TODO parallelization without mod operator (mod is slow on GPU)
    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_multiplyColumnsByScalars <<< gridSize , THREADS_PER_BLOCK >>> (vectorDevPtr, matrixDevPtr, nRows, nElements);
}

template __global__ void __cuda_multiplyColumnsByScalars<double>(const double *vectorDevPtr, double *matrixDevPtr, unsigned int nRows, unsigned int nElements);
template void _cuda_multiplyColumnsByScalars<double>(const double *vectorDevPtr, double *matrixDevPtr, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_multiplyColumnsByScalars<float>(const float *vectorDevPtr, float *matrixDevPtr, unsigned int nRows, unsigned int nElements);
template void _cuda_multiplyColumnsByScalars<float>(const float *vectorDevPtr, float *matrixDevPtr, unsigned int nRows, unsigned int nColumns);

/*
 *
 * divideColumnsByScalars
 *
 */
template<typename T>
__global__ void __cuda_divideColumnsByScalars(const T *vectorDevPtr, T *matrixDevPtr, unsigned int nRows, unsigned int nElements){
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int colIndex = index / nRows;
    if (index < nElements)
	matrixDevPtr[index] = matrixDevPtr[index] / vectorDevPtr[colIndex];
}
template<typename T>
void _cuda_divideColumnsByScalars(const T *vectorDevPtr, T *matrixDevPtr, unsigned int nRows, unsigned int nColumns){
    // TODO parallelization without mod operator (mod is slow on GPU)
    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_divideColumnsByScalars <<< gridSize , THREADS_PER_BLOCK >>> (vectorDevPtr, matrixDevPtr, nRows, nElements);
}

template __global__ void __cuda_divideColumnsByScalars<double>(const double *vectorDevPtr, double *matrixDevPtr, unsigned int nRows, unsigned int nElements);
template void _cuda_divideColumnsByScalars<double>(const double *vectorDevPtr, double *matrixDevPtr, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_divideColumnsByScalars<float>(const float *vectorDevPtr, float *matrixDevPtr, unsigned int nRows, unsigned int nElements);
template void _cuda_divideColumnsByScalars<float>(const float *vectorDevPtr, float *matrixDevPtr, unsigned int nRows, unsigned int nColumns);

/*
 *
 * multiplyRowsByScalars
 *
 */
template<typename T>
__global__ void __cuda_multiplyRowsByScalars(const T *vectorDevPtr, T *matrixDevPtr, unsigned int nRows, unsigned int nElements){
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int rowIndex = index % nRows;
    if (index < nElements)
	matrixDevPtr[index] = matrixDevPtr[index] * vectorDevPtr[rowIndex];
}
template<typename T>
void _cuda_multiplyRowsByScalars(const T *vectorDevPtr, T *matrixDevPtr, unsigned int nRows, unsigned int nColumns){
    // TODO parallelization without mod operator (mod is slow on GPU)
    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_multiplyRowsByScalars <<< gridSize , THREADS_PER_BLOCK >>> (vectorDevPtr, matrixDevPtr, nRows, nElements);
}


template __global__ void __cuda_multiplyRowsByScalars<double>(const double *vectorDevPtr, double *matrixDevPtr, unsigned int nRows,unsigned int nElements);
template void _cuda_multiplyRowsByScalars<double>(const double *vectorDevPtr, double *matrixDevPtr, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_multiplyRowsByScalars<float>(const float *vectorDevPtr, float *matrixDevPtr, unsigned int nRows, unsigned int nElements);
template void _cuda_multiplyRowsByScalars<float>(const float *vectorDevPtr, float *matrixDevPtr, unsigned int nRows, unsigned int nColumns);

/*
 *
 * divideRowsByScalars
 *
 */
template<typename T>
__global__ void __cuda_divideRowsByScalars(const T *vectorDevPtr, T *matrixDevPtr, unsigned int nRows, unsigned int nElements){
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int rowIndex = index % nRows;
    if (index < nElements)
	matrixDevPtr[index] = matrixDevPtr[index] / vectorDevPtr[rowIndex];
}
template<typename T>
void _cuda_divideRowsByScalars(const T *vectorDevPtr, T *matrixDevPtr, unsigned int nRows, unsigned int nColumns){
    // TODO parallelization without mod operator (mod is slow on GPU)
    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_divideRowsByScalars <<< gridSize , THREADS_PER_BLOCK >>> (vectorDevPtr, matrixDevPtr, nRows, nElements);
}


template __global__ void __cuda_divideRowsByScalars<double>(const double *vectorDevPtr, double *matrixDevPtr, unsigned int nRows,unsigned int nElements);
template void _cuda_divideRowsByScalars<double>(const double *vectorDevPtr, double *matrixDevPtr, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_divideRowsByScalars<float>(const float *vectorDevPtr, float *matrixDevPtr, unsigned int nRows,unsigned int nElements);
template void _cuda_divideRowsByScalars<float>(const float *vectorDevPtr, float *matrixDevPtr, unsigned int nRows, unsigned int nColumns);

/*
 *
 *  fill
 *
 */
template<typename T>
__global__ void __cuda_fill(T *data, T value, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
	data[index] = value;
}
template<typename T>
void _cuda_fill(T *data, T value, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_fill <<< gridSize , THREADS_PER_BLOCK >>> (data, value, nElements);
}

template __global__ void __cuda_fill<double>(double *data, double value, unsigned int nElements);
template void _cuda_fill<double>(double *data, double value, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_fill<float>(float *data, float value, unsigned int nElements);
template void _cuda_fill<float>(float *data, float value, unsigned int nRows, unsigned int nColumns);

/*
 *
 * Average Pooling
 *
 */
template<typename T>
__global__ void __cuda_avgPool(const T *source, T *result, const unsigned int sourceRows, 
	const unsigned int sourceColumns, const unsigned int sourceWidth, const unsigned int sourceHeight, 
	const unsigned int sourceChannels, const unsigned int poolSize, const unsigned int stride, const T minValue)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	unsigned int resultWidth = ceil((float)sourceWidth/stride);
	unsigned int resultHeight = ceil((float)sourceHeight/stride); 
	
	unsigned int resultRows = resultWidth * resultHeight * sourceChannels;
	
	if(index < (resultRows * sourceColumns))
	{
		int imageNum = index / resultRows;
		int resultPixelNum = index % resultRows;
		
		int channelNum = resultPixelNum / (resultWidth * resultHeight);
		resultPixelNum %= (resultWidth * resultHeight);
		
		int resultPixelX = resultPixelNum / resultHeight;
		int resultPixelY = resultPixelNum % resultHeight;
		
		int sourcePixelX = resultPixelX * stride;
		int sourcePixelY = resultPixelY * stride;
		
		T sum = 0;
		T num = 0;
		int index = -1;
		for(int i=sourcePixelX; (i<(sourcePixelX+poolSize)) && (i<sourceWidth); i++)
		{		
			for(int j=sourcePixelY; (j<(sourcePixelY + poolSize)) && (j<sourceHeight); j++)
			{	
				index = imageNum * sourceRows + channelNum * (sourceWidth * sourceHeight) + 
				i * sourceHeight + j;
				sum +=  source[index];
				num += 1;
			}
		}
		int resultIndex = imageNum * resultRows + channelNum * (resultWidth * resultHeight) + 
			resultPixelX * resultHeight + resultPixelY;
		result[resultIndex] = sum / (poolSize * poolSize);//num;
	}
}
template<typename T>
void _cuda_avgPool(const T *source, T *result, const unsigned int sourceRows,
	const unsigned int sourceColumns, const unsigned int sourceWidth, const unsigned int sourceHeight,
	const unsigned int sourceChannels, const unsigned int poolSize, const unsigned int stride)
{
	unsigned int nElements = (int)ceil((float)sourceWidth/stride) * 
		(int)ceil((float)sourceHeight/stride) * sourceChannels * sourceColumns;
		
	int gridSize = (int)ceil((float) nElements/THREADS_PER_BLOCK);
	
	__cuda_avgPool<<<gridSize, THREADS_PER_BLOCK>>>(source, result, sourceRows,
	 sourceColumns, sourceWidth, sourceHeight, sourceChannels, poolSize, stride, std::numeric_limits<T>::min());
}

template __global__ void __cuda_avgPool(const double *source, double *result, 
	const unsigned int sourceRows, const unsigned int sourceColumns, const unsigned int sourceWidth, 
	const unsigned int sourceHeight, const unsigned int sourceChannels, const unsigned int poolSize, 
	const unsigned int stride, double minValue);
template __global__ void __cuda_avgPool(const float *source, float *result, 
	const unsigned int sourceRows, const unsigned int sourceColumns, const unsigned int sourceWidth, 
	const unsigned int sourceHeight, const unsigned int sourceChannels, const unsigned int poolSize, 
	const unsigned int stride, float minValue);
template void _cuda_avgPool(const double *source, double *result, const unsigned int sourceRows, 
	const unsigned int sourceColumns, const unsigned int sourceWidth, const unsigned int sourceHeight, 
	const unsigned int sourceChannels, const unsigned int poolSize, const unsigned int stride);
template void _cuda_avgPool(const float *source, float *result, const unsigned int sourceRows, 
	const unsigned int sourceColumns, const unsigned int sourceWidth, const unsigned int sourceHeight, 
	const unsigned int sourceChannels, const unsigned int poolSize, const unsigned int stride);

/*
 *
 * Avg Pooling Backpropogation 
 *
 */
		
template<typename T>
__global__ void __cuda_backPropogateAvgPool(T *result, const T *errorSignal, const unsigned int sourceRows,
		const unsigned int sourceColumns, const unsigned int sourceWidth, const unsigned int sourceHeight,
		const unsigned int sourceChannels, const unsigned int poolSize, const unsigned int stride)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	unsigned int errorSignalWidth = ceil((double)sourceWidth/stride);
	unsigned int errorSignalHeight = ceil((double)sourceHeight/stride); 
	
	unsigned int errorSignalRows = errorSignalWidth * errorSignalHeight * sourceChannels;
	
	
	if(index < (sourceRows * sourceColumns))
	{
		int imageNum = index / sourceRows;
		int imagePixelNum = index % sourceRows;
		
		int channel = imagePixelNum / (sourceWidth * sourceHeight);
		imagePixelNum %=  (sourceWidth * sourceHeight);
		
		int pixelX = imagePixelNum / sourceHeight;
		int pixelY = imagePixelNum % sourceHeight;
		
		int indexInErrorSignal = -1;

		//calculates start of the first grid containing current Pixel
		unsigned int gridStartX = (pixelX + 1 - (int)poolSize) < 0 ? 0 :
			(unsigned int)(ceil((float)(pixelX + 1 - (int)poolSize)/(float)stride) * stride);
		unsigned int gridStartY = (pixelY + 1 - (int)poolSize) < 0 ? 0 :
			(unsigned int)(ceil((float)(pixelY + 1 - (int)poolSize)/(float)stride) * stride);
		//////////////////////////////////

		for(unsigned int gridX=gridStartX; gridX<=pixelX; gridX+=stride)
		{  
			for(unsigned int gridY=gridStartY; gridY<=pixelY; gridY+=stride)
			{
				indexInErrorSignal = imageNum * errorSignalRows + channel * errorSignalHeight * errorSignalWidth +
					(gridX/stride) * errorSignalHeight + (gridY/stride);
				
				result[index] += errorSignal[indexInErrorSignal] / (T)(poolSize * poolSize);				
			}
		}   
	}
}

template<typename T>
void _cuda_backPropogateAvgPool(T *result, const T *errorSignal,
		const unsigned int sourceRows, const unsigned int sourceColumns, const unsigned int sourceWidth, 
		const unsigned int sourceHeight, const unsigned int sourceChannels, const unsigned int poolSize, 
		const unsigned int stride)
{
	unsigned int nElements = sourceRows * sourceColumns;
		
	int gridSize = (int)ceil((float) nElements/THREADS_PER_BLOCK);
	
	__cuda_backPropogateAvgPool<<<gridSize, THREADS_PER_BLOCK>>>(result, errorSignal, 
	sourceRows, sourceColumns,	sourceWidth, sourceHeight, sourceChannels, poolSize, stride);
}

template __global__ void __cuda_backPropogateAvgPool(double *result, const double *errorSignal,
		const unsigned int sourceRows, const unsigned int sourceColumns, const unsigned int sourceWidth,
		const unsigned int sourceHeight, const unsigned int sourceChannels, const unsigned int poolSize,
		const unsigned int stride);
template __global__ void __cuda_backPropogateAvgPool(float *result, const float *errorSignal,
		const unsigned int sourceRows, const unsigned int sourceColumns, const unsigned int sourceWidth,
		const unsigned int sourceHeight, const unsigned int sourceChannels, const unsigned int poolSize,
		const unsigned int stride);
		
template void _cuda_backPropogateAvgPool(double *result, const double *errorSignal,
		const unsigned int sourceRows, const unsigned int sourceColumns, const unsigned int sourceWidth,
		const unsigned int sourceHeight, const unsigned int sourceChannels, const unsigned int poolSize,
		const unsigned int stride);
template void _cuda_backPropogateAvgPool(float *result, const float *errorSignal,
		const unsigned int sourceRows, const unsigned int sourceColumns, const unsigned int sourceWidth,
		const unsigned int sourceHeight, const unsigned int sourceChannels, const unsigned int poolSize,
		const unsigned int stride);




/*
 *
 * Max Pooling
 *
 */
template<typename T>
__global__ void __cuda_maxPool(const T *source, T *result, const unsigned int sourceRows, 
	const unsigned int sourceColumns, const unsigned int sourceWidth, const unsigned int sourceHeight, 
	const unsigned int sourceChannels, const unsigned int poolSize, const unsigned int stride, const T minValue)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	unsigned int resultWidth = ceil((float)sourceWidth/stride);
	unsigned int resultHeight = ceil((float)sourceHeight/stride); 
	
	unsigned int resultRows = resultWidth * resultHeight * sourceChannels;
	
	if(index < (resultRows * sourceColumns))
	{
		int imageNum = index / resultRows;
		int resultPixelNum = index % resultRows;
		
		int channelNum = resultPixelNum / (resultWidth * resultHeight);
		resultPixelNum %= (resultWidth * resultHeight);
		
		int resultPixelX = resultPixelNum / resultHeight;
		int resultPixelY = resultPixelNum % resultHeight;
		
		int sourcePixelX = resultPixelX * stride;
		int sourcePixelY = resultPixelY * stride;
		
		T maxValue = minValue;
		int index = -1;
		for(int i=sourcePixelX; (i<(sourcePixelX+poolSize)) && (i<sourceWidth); i++)
		{		
			for(int j=sourcePixelY; (j<(sourcePixelY + poolSize)) && (j<sourceHeight); j++)
			{	
				index = imageNum * sourceRows + channelNum * (sourceWidth * sourceHeight) + 
				i * sourceHeight + j; 
				if(source[index] >= maxValue)
				{
					maxValue = source[index]; 
				}
			}
		}
		int resultIndex = imageNum * resultRows + channelNum * (resultWidth * resultHeight) + 
			resultPixelX * resultHeight + resultPixelY;
		result[resultIndex] = maxValue;
	}
}
template<typename T>
void _cuda_maxPool(const T *source, T *result, const unsigned int sourceRows,
	const unsigned int sourceColumns, const unsigned int sourceWidth, const unsigned int sourceHeight,
	const unsigned int sourceChannels, const unsigned int poolSize, const unsigned int stride)
{
	unsigned int nElements = (int)ceil((float)sourceWidth/stride) * 
		(int)ceil((float)sourceHeight/stride) * sourceChannels * sourceColumns;
		
	int gridSize = (int)ceil((float) nElements/THREADS_PER_BLOCK);
	
	__cuda_maxPool<<<gridSize, THREADS_PER_BLOCK>>>(source, result, sourceRows,
	 sourceColumns, sourceWidth, sourceHeight, sourceChannels, poolSize, stride, std::numeric_limits<T>::min());
}

template __global__ void __cuda_maxPool(const double *source, double *result, 
	const unsigned int sourceRows, const unsigned int sourceColumns, const unsigned int sourceWidth, 
	const unsigned int sourceHeight, const unsigned int sourceChannels, const unsigned int poolSize, 
	const unsigned int stride, double minValue);
template __global__ void __cuda_maxPool(const float *source, float *result, 
	const unsigned int sourceRows, const unsigned int sourceColumns, const unsigned int sourceWidth, 
	const unsigned int sourceHeight, const unsigned int sourceChannels, const unsigned int poolSize, 
	const unsigned int stride, float minValue);
template void _cuda_maxPool(const double *source, double *result, const unsigned int sourceRows, 
	const unsigned int sourceColumns, const unsigned int sourceWidth, const unsigned int sourceHeight, 
	const unsigned int sourceChannels, const unsigned int poolSize, const unsigned int stride);
template void _cuda_maxPool(const float *source, float *result, const unsigned int sourceRows, 
	const unsigned int sourceColumns, const unsigned int sourceWidth, const unsigned int sourceHeight, 
	const unsigned int sourceChannels, const unsigned int poolSize, const unsigned int stride);


/*
 *
 * Max Pooling Backpropogation 
 *
 */
		
template<typename T>
__global__ void __cuda_backPropogateMaxPool(T *result, const T* activationIn, const T* activationOut, const T *errorSignal,const unsigned int sourceRows,
		const unsigned int sourceColumns, const unsigned int sourceWidth, const unsigned int sourceHeight,
		const unsigned int sourceChannels, const unsigned int poolSize, const unsigned int stride)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	unsigned int errorSignalWidth = ceil((double)sourceWidth/stride);
	unsigned int errorSignalHeight = ceil((double)sourceHeight/stride); 
	
	unsigned int errorSignalRows = errorSignalWidth * errorSignalHeight * sourceChannels;
	
	
	if(index < (sourceRows * sourceColumns))
	{
		int imageNum = index / sourceRows;
		int imagePixelNum = index % sourceRows;
		
		int channel = imagePixelNum / (sourceWidth * sourceHeight);
		imagePixelNum %=  (sourceWidth * sourceHeight);
		
		int pixelX = imagePixelNum / sourceHeight;
		int pixelY = imagePixelNum % sourceHeight;
		
		int indexInErrorSignal = -1;
		
		int numMaxima = 0;


		//calculates start of the first grid containing current Pixel
		unsigned int gridStartX = (pixelX + 1 - (int)poolSize) < 0 ? 0 :
			(unsigned int)(ceil((float)(pixelX + 1 - (int)poolSize)/(float)stride) * stride);
		unsigned int gridStartY = (pixelY + 1 - (int)poolSize) < 0 ? 0 :
			(unsigned int)(ceil((float)(pixelY + 1 - (int)poolSize)/(float)stride) * stride);
		//////////////////////////////////

		for(unsigned int gridX=gridStartX; gridX<=pixelX; gridX+=stride)
		{
			for(unsigned int gridY=gridStartY; gridY<=pixelY; gridY+=stride)
			{
				indexInErrorSignal = imageNum * errorSignalRows + channel * errorSignalHeight * errorSignalWidth +
					(gridX/stride) * errorSignalHeight + (gridY/stride);

				//current pixel is not maximum in current window
				if(activationIn[index] != activationOut[indexInErrorSignal])
					break;
					
				numMaxima = 0;
				for(unsigned int i=gridX; (i<(gridX + poolSize)) && i<sourceWidth; i++)
				{
					for(unsigned int j=gridY;(j<(gridY+poolSize)) && j<sourceHeight; j++)
					{
						int indexInActivationIn = imageNum * sourceRows + channel * sourceHeight * sourceWidth +
							i * sourceHeight + j;
						if(activationIn[indexInActivationIn] ==
							activationOut[indexInErrorSignal])
						{
							numMaxima += 1;
						}
					}
				}
				result[index] += errorSignal[indexInErrorSignal] / (T) numMaxima;				
			}
		}   
	}
}

template<typename T>
void _cuda_backPropogateMaxPool(T *result, const T* activationIn, const T* activationOut, const T *errorSignal,
		const unsigned int sourceRows, const unsigned int sourceColumns, const unsigned int sourceWidth, 
		const unsigned int sourceHeight, const unsigned int sourceChannels, const unsigned int poolSize, 
		const unsigned int stride)
{
	unsigned int nElements = sourceRows * sourceColumns;
		
	int gridSize = (int)ceil((float) nElements/THREADS_PER_BLOCK);
	
	__cuda_backPropogateMaxPool<<<gridSize, THREADS_PER_BLOCK>>>(result, activationIn, activationOut, 
		errorSignal, sourceRows, sourceColumns,	sourceWidth, sourceHeight, sourceChannels, poolSize, stride);
}

template __global__ void __cuda_backPropogateMaxPool(double *result, const double* activationIn, const double* activationOut, const double *errorSignal,
		const unsigned int sourceRows, const unsigned int sourceColumns, const unsigned int sourceWidth,
		const unsigned int sourceHeight, const unsigned int sourceChannels, const unsigned int poolSize,
		const unsigned int stride);
template __global__ void __cuda_backPropogateMaxPool(float *result, const float* activationIn, const float* activationOut, const float *errorSignal,
		const unsigned int sourceRows, const unsigned int sourceColumns, const unsigned int sourceWidth,
		const unsigned int sourceHeight, const unsigned int sourceChannels, const unsigned int poolSize,
		const unsigned int stride);
		
template void _cuda_backPropogateMaxPool(double *result, const double* activationIn, const double* activationOut, const double *errorSignal,
		const unsigned int sourceRows, const unsigned int sourceColumns, const unsigned int sourceWidth,
		const unsigned int sourceHeight, const unsigned int sourceChannels, const unsigned int poolSize,
		const unsigned int stride);
template void _cuda_backPropogateMaxPool(float *result, const float* activationIn, const float* activationOut, const float *errorSignal,
		const unsigned int sourceRows, const unsigned int sourceColumns, const unsigned int sourceWidth,
		const unsigned int sourceHeight, const unsigned int sourceChannels, const unsigned int poolSize,
		const unsigned int stride);



/*
 *
 *  ensure minimal value
 *
 */
template<typename T>
__global__ void __cuda_ensureMinimalValue(T *data, T value, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if ((index < nElements) && (data[index] < value))
	data[index] = value;
}

template<typename T>
void _cuda_ensureMinimalValue(T *data, T value, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_ensureMinimalValue <<< gridSize , THREADS_PER_BLOCK >>> (data, value, nElements);
}

template __global__ void __cuda_ensureMinimalValue(double *data, double value, unsigned int nElements);
template void _cuda_ensureMinimalValue(double *data, double value, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_ensureMinimalValue(float *data, float value, unsigned int nElements);
template void _cuda_ensureMinimalValue(float *data, float value, unsigned int nRows, unsigned int nColumns);

/*
 *
 *  ensure maximal value
 *
 */
template<typename T>
__global__ void __cuda_ensureMaximalValue(T *data, T value, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if ((index < nElements) && (data[index] > value))
	data[index] = value;
}

template<typename T>
void _cuda_ensureMaximalValue(T *data, T value, unsigned int nRows, unsigned int nColumns)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_ensureMaximalValue <<< gridSize , THREADS_PER_BLOCK >>> (data, value, nElements);
}

template __global__ void __cuda_ensureMaximalValue(double *data, double value, unsigned int nElements);
template void _cuda_ensureMaximalValue(double *data, double value, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_ensureMaximalValue(float *data, float value, unsigned int nElements);
template void _cuda_ensureMaximalValue(float *data, float value, unsigned int nRows, unsigned int nColumns);

/*
 *
 * prepares for convolution
 *
 */
template<typename T>
__global__ void __cuda_prepareConvolution(T* dest, const T* source, const unsigned int sourceWidth, const unsigned int sourceHeight,
		const unsigned int sourceChannels, const unsigned int kernelWidth, const unsigned int kernelHeight,
		const unsigned int destRows, const unsigned int destCols, const unsigned int strideX, const unsigned int strideY)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < destRows * destCols) {
		int imageNum = index / destRows;
		int resultIndex = index % destRows;
		int kernelMiddleX = kernelWidth / 2;
		int kernelMiddleY = kernelHeight / 2;
		int heightOfOneDestCh = (int)ceil((float)(sourceHeight - kernelHeight + 1) / (float)strideY);
		
		int pixelNum = resultIndex / (kernelHeight * kernelWidth * sourceChannels);
		int pixelX = (pixelNum / heightOfOneDestCh) * strideX + kernelMiddleX;
		int pixelY = (pixelNum % heightOfOneDestCh) * strideY + kernelMiddleY;
		int channelNum = resultIndex % (kernelHeight * kernelWidth * sourceChannels);
		int neighbNum = channelNum % (kernelHeight * kernelWidth);
		channelNum = channelNum / (kernelWidth * kernelHeight);
		int neighX = (neighbNum / kernelHeight) - kernelMiddleX;
		int neighY = (neighbNum % kernelHeight) - kernelMiddleY;
		
		dest[index] = source[imageNum * (sourceChannels * sourceWidth * sourceHeight) + 
						channelNum * (sourceWidth * sourceHeight) + 
						(pixelX + neighX) * sourceHeight + (pixelY + neighY)];
	}
}
template<typename T>
void _cuda_prepareConvolution(T* dest, const T* source, const unsigned int sourceWidth, const unsigned int sourceHeight,
		const unsigned int sourceChannels, const unsigned int kernelWidth, const unsigned int kernelHeight,
		const unsigned int destRows, const unsigned int destCols, const unsigned int strideX, const unsigned int strideY)
{
	unsigned int nElements = destRows * destCols;
	int gridSize = (int)ceil((float)nElements/ THREADS_PER_BLOCK);
	
	__cuda_prepareConvolution<<<gridSize, THREADS_PER_BLOCK>>>(dest, source, sourceWidth, sourceHeight,
		sourceChannels, kernelWidth, kernelHeight, destRows, destCols, strideX, strideY);
}
template __global__ void __cuda_prepareConvolution(double* dest, const double* source, const unsigned int sourceWidth, const unsigned int sourceHeight,
		const unsigned int sourceChannels, const unsigned int kernelWidth, const unsigned int kernelHeight,
		const unsigned int destRows, const unsigned int destCols, const unsigned int strideX, const unsigned int strideY);
template __global__ void __cuda_prepareConvolution(float* dest, const float* source, const unsigned int sourceWidth, const unsigned int sourceHeight,
		const unsigned int sourceChannels, const unsigned int kernelWidth, const unsigned int kernelHeight,
		const unsigned int destRows, const unsigned int destCols, const unsigned int strideX, const unsigned int strideY);
		
template void _cuda_prepareConvolution(double* dest, const double* source, const unsigned int sourceWidth, const unsigned int sourceHeight,
		const unsigned int sourceChannels, const unsigned int kernelWidth, const unsigned int kernelHeight,
		const unsigned int destRows, const unsigned int destCols, const unsigned int strideX, const unsigned int strideY);
template void _cuda_prepareConvolution(float* dest, const float* source, const unsigned int sourceWidth, const unsigned int sourceHeight,
		const unsigned int sourceChannels, const unsigned int kernelWidth, const unsigned int kernelHeight,
		const unsigned int destRows, const unsigned int destCols, const unsigned int strideX, const unsigned int strideY);
/*
 * Backpropogation convolution
 *
 */
template<typename T>
__global__ void __cuda_prepareConvolutionBackProp(T* dest, const T* source, const unsigned int destWidth, 
	const unsigned int destHeight, const unsigned int destChannels, const unsigned int kernelWidth, 
	const unsigned int kernelHeight, const unsigned int destRows, const unsigned int destCols)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < destRows * destCols) {
		dest[index] = 0;
		int img = index / destRows;
		int ch = (index % destRows) / (destWidth * destHeight);
		int pixelNum = (index % destRows) % (destWidth * destHeight);
		
		int pixelX = pixelNum / destHeight;
		int pixelY = pixelNum % destHeight;
		
		int gridStartX = (pixelX + 1 - (int)kernelWidth) <= 0 ? 0 : 
			(pixelX + 1 - (int)kernelWidth);
		int gridStartY = (pixelY + 1 - (int)kernelHeight) <= 0 ? 0 :
					(pixelY + 1 - (int)kernelHeight);
		int sourceHeight = (destHeight - (int)kernelHeight + 1);
		int sizeOfOneChSource = sourceHeight * (destWidth - (int)kernelWidth + 1);
		int neighNum = 0;
		for(int j=gridStartX; (j<=pixelX) && ((j + kernelWidth) <= destWidth); j++) {
			for(int k=gridStartY; (k<=pixelY) && ((k + kernelHeight) <= destHeight) ; k++) {
				// (Cx, Cy) = (j + kernelMiddleX, k + kernelMiddleY) are coordinates of center pixel in grid
				// (Rx, Ry) = (Cx - pixelX, Cy - pixelY) gives coordinates of pixel in refernce
				// to center pixel, such that center pixel of grid is mapped is mapped to (0,0)
				neighNum = (pixelX - j) * kernelHeight + (pixelY - k);
				//(j * sourceHeight + k) is pixel number of center of grid in source
				//i.e result of convolution
				dest[index] += source[img * sizeOfOneChSource * destChannels * kernelWidth * kernelHeight + 
									(j * sourceHeight + k) * destChannels * kernelWidth * kernelHeight +
									ch * kernelWidth * kernelHeight + neighNum];
			}
		}
	}
}
template<typename T>
void _cuda_prepareConvolutionBackProp(T* dest, const T* source, const unsigned int destWidth, 
	const unsigned int destHeight, const unsigned int destChannels, const unsigned int kernelWidth, 
	const unsigned int kernelHeight, const unsigned int destRows, const unsigned int destCols)
{
	unsigned int nElements = destRows * destCols;
	int gridSize = (int)ceil((float)nElements/ THREADS_PER_BLOCK);
	
	__cuda_prepareConvolutionBackProp<<<gridSize, THREADS_PER_BLOCK>>>(dest, source, destWidth, destHeight,
		destChannels, kernelWidth, kernelHeight, destRows, destCols);
}
template void _cuda_prepareConvolutionBackProp(double* dest, const double* source, const unsigned int destWidth, 
	const unsigned int destHeight, const unsigned int destChannels, const unsigned int kernelWidth, 
	const unsigned int kernelHeight, const unsigned int destRows, const unsigned int destCols);
template void _cuda_prepareConvolutionBackProp(float* dest, const float* source, const unsigned int destWidth, 
	const unsigned int destHeight, const unsigned int destChannels, const unsigned int kernelWidth, 
	const unsigned int kernelHeight, const unsigned int destRows, const unsigned int destCols);
template __global__ void __cuda_prepareConvolutionBackProp(double* dest, const double* source, const unsigned int destWidth, 
	const unsigned int destHeight, const unsigned int destChannels, const unsigned int kernelWidth, 
	const unsigned int kernelHeight, const unsigned int destRows, const unsigned int destCols);	
template __global__ void __cuda_prepareConvolutionBackProp(float* dest, const float* source, const unsigned int destWidth, 
	const unsigned int destHeight, const unsigned int destChannels, const unsigned int kernelWidth, 
	const unsigned int kernelHeight, const unsigned int destRows, const unsigned int destCols);
/*
 * prepare for convolution such that after convolution image size stays same
 *
 *
 */
template<typename T>
__global__ void __cuda_prepareConvolutionSame(T* dest, const T* source, const unsigned int sourceWidth, const unsigned int sourceHeight,
		const unsigned int sourceChannels, const unsigned int kernelWidth, const unsigned int kernelHeight,
		const unsigned int destRows, const unsigned int destCols, const unsigned int strideX, const unsigned int strideY)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < destRows * destCols) {
		int destWidth = (int)ceil((float)sourceWidth / (float)strideX);
		int destHeight = (int)ceil((float)sourceHeight / (float)strideY);
		
		int imageNum = index / destRows;
		int resultIndex = index % destRows;
		int kernelMiddleX = kernelWidth / 2;
		int kernelMiddleY = kernelHeight / 2;
		
		
		int pixelNum = resultIndex / (kernelHeight * kernelWidth * sourceChannels);
		
		int pixelX = (pixelNum / destHeight) * strideX;
		int pixelY = (pixelNum % destHeight) * strideY;
		
		int channelNum = resultIndex % (kernelHeight * kernelWidth * sourceChannels);
		int neighbNum = channelNum % (kernelHeight * kernelWidth);
		channelNum = channelNum / (kernelWidth * kernelHeight);
		int neighX = (neighbNum / kernelHeight) - kernelMiddleX;
		int neighY = (neighbNum % kernelHeight) - kernelMiddleY;
		
		dest[index] = ( (pixelX + neighX) < 0 || (pixelY + neighY) < 0 || 
						(pixelX + neighX) >= sourceWidth || (pixelY + neighY) >= sourceHeight) ? 0 : 
					source[imageNum * (sourceChannels * sourceWidth * sourceHeight) + 
						channelNum * (sourceWidth * sourceHeight) + 
							(pixelX + neighX) * sourceHeight + (pixelY + neighY)];
	}
}
template<typename T>
void _cuda_prepareConvolutionSame(T* dest, const T* source, const unsigned int sourceWidth, const unsigned int sourceHeight,
		const unsigned int sourceChannels, const unsigned int kernelWidth, const unsigned int kernelHeight,
		const unsigned int destRows, const unsigned int destCols, const unsigned int strideX, const unsigned int strideY)
{
	unsigned int nElements = destRows * destCols;
	int gridSize = (int)ceil((float)nElements/ THREADS_PER_BLOCK);
	
	__cuda_prepareConvolutionSame<<<gridSize, THREADS_PER_BLOCK>>>(dest, source, sourceWidth, sourceHeight,
		sourceChannels, kernelWidth, kernelHeight, destRows, destCols, strideX, strideY);
}
template __global__ void __cuda_prepareConvolutionSame(double* dest, const double* source, const unsigned int sourceWidth, const unsigned int sourceHeight,
		const unsigned int sourceChannels, const unsigned int kernelWidth, const unsigned int kernelHeight,
		const unsigned int destRows, const unsigned int destCols, const unsigned int strideX, const unsigned int strideY);
template __global__ void __cuda_prepareConvolutionSame(float* dest, const float* source, const unsigned int sourceWidth, const unsigned int sourceHeight,
		const unsigned int sourceChannels, const unsigned int kernelWidth, const unsigned int kernelHeight,
		const unsigned int destRows, const unsigned int destCols, const unsigned int strideX, const unsigned int strideY);
		
template void _cuda_prepareConvolutionSame(double* dest, const double* source, const unsigned int sourceWidth, const unsigned int sourceHeight,
		const unsigned int sourceChannels, const unsigned int kernelWidth, const unsigned int kernelHeight,
		const unsigned int destRows, const unsigned int destCols, const unsigned int strideX, const unsigned int strideY);
template void _cuda_prepareConvolutionSame(float* dest, const float* source, const unsigned int sourceWidth, const unsigned int sourceHeight,
		const unsigned int sourceChannels, const unsigned int kernelWidth, const unsigned int kernelHeight,
		const unsigned int destRows, const unsigned int destCols, const unsigned int strideX, const unsigned int strideY);

template<typename T>
__global__ void __cuda_prepareConvolutionSameBackProp(T* dest, const T* source, const unsigned int destWidth, 
	const unsigned int destHeight, const unsigned int destChannels, const unsigned int kernelWidth, 
	const unsigned int kernelHeight, const unsigned int destRows, const unsigned int destCols, 
	const unsigned int strideX, const unsigned int strideY)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < (destRows * destCols)) {
		dest[index] = 0;
		int img = index / destRows; // destRows = destWidth * destHeight * destChannels
		int ch = (index % destRows) / (destWidth * destHeight);
		int pixelNum = (index % destRows) % (destWidth * destHeight);
		
		int pixelX = pixelNum / destHeight;
		int pixelY = pixelNum % destHeight;
		
		int kernelMiddleX = (int)kernelWidth / 2;
		int kernelMiddleY = (int)kernelHeight / 2;
		
		int gridStartX = (pixelX + 1 - (int)kernelWidth) <= (-1 * kernelMiddleX) ? (-1 * kernelMiddleX) :
			(pixelX + 1 - (int)kernelWidth);
		int gridStartY = (pixelY + 1 - (int)kernelHeight) <= (-1 * kernelMiddleY) ? (-1 * kernelMiddleY) :
			(pixelY + 1 - (int)kernelHeight);

		for(int gridX=gridStartX; (gridX <= pixelX) && ((gridX + kernelMiddleX) < destWidth) ; gridX++) {
			if (((gridX + kernelMiddleX) % strideX) == 0) {
				for(int gridY=gridStartY; (gridY <= pixelY) && ((gridY + kernelMiddleY) < destHeight); gridY++) {
					if (((gridY + kernelMiddleY) % strideY) == 0) {
						int neighNum = (pixelX - gridX) * kernelHeight + (pixelY - gridY);
						int centerPixel = (((gridX + kernelMiddleX) / strideX) * destHeight / strideY) + (gridY + kernelMiddleY) / strideY;
						
						dest[index] += source[img * destChannels * (destWidth / strideX) * (destHeight / strideY) * kernelWidth * kernelHeight + 
							centerPixel * destChannels * kernelWidth * kernelHeight + ch * kernelWidth * kernelHeight + neighNum];					
					}		
				}				
			}
		}
	}
}
template<typename T>
void _cuda_prepareConvolutionSameBackProp(T* dest, const T* source, const unsigned int destWidth, 
	const unsigned int destHeight, const unsigned int destChannels, const unsigned int kernelWidth, 
	const unsigned int kernelHeight, const unsigned int destRows, const unsigned int destCols,
	const unsigned int strideX, const unsigned int strideY)
{
	unsigned int nElements = destRows * destCols;
	int gridSize = (int)ceil((float)nElements/ THREADS_PER_BLOCK);
	
	__cuda_prepareConvolutionSameBackProp<<<gridSize, THREADS_PER_BLOCK>>>(dest, source, destWidth, destHeight,
		destChannels, kernelWidth, kernelHeight, destRows, destCols, strideX, strideY);
}
template void _cuda_prepareConvolutionSameBackProp(double* dest, const double* source, const unsigned int destWidth, 
	const unsigned int destHeight, const unsigned int destChannels, const unsigned int kernelWidth, 
	const unsigned int kernelHeight, const unsigned int destRows, const unsigned int destCols, 
	const unsigned int strideX, const unsigned int strideY);
	
template void _cuda_prepareConvolutionSameBackProp(float* dest, const float* source, const unsigned int destWidth, 
	const unsigned int destHeight, const unsigned int destChannels, const unsigned int kernelWidth, 
	const unsigned int kernelHeight, const unsigned int destRows, const unsigned int destCols, 
	const unsigned int strideX, const unsigned int strideY);
	
template __global__ void __cuda_prepareConvolutionSameBackProp(double* dest, const double* source, const unsigned int destWidth, 
	const unsigned int destHeight, const unsigned int destChannels, const unsigned int kernelWidth, 
	const unsigned int kernelHeight, const unsigned int destRows, const unsigned int destCols, 
	const unsigned int strideX, const unsigned int strideY);
	
template __global__ void __cuda_prepareConvolutionSameBackProp(float* dest, const float* source, const unsigned int destWidth, 
	const unsigned int destHeight, const unsigned int destChannels, const unsigned int kernelWidth, 
	const unsigned int kernelHeight, const unsigned int destRows, const unsigned int destCols, 
	const unsigned int strideX, const unsigned int strideY);
/*
 * rearrange 
 *
 * helper for convolution 
 */	
		
template<typename T>
__global__ void __cuda_rearrange(T *dest, const T *source, const unsigned int sourceRows,
		const unsigned int destRows, const unsigned int destColumns, const unsigned int destNumPixels)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < (destRows * destColumns)) {
		unsigned int img = index / (sourceRows * destNumPixels);
		unsigned int ch = (index % (sourceRows * destNumPixels)) / destNumPixels;
		unsigned int pix = (index % (sourceRows * destNumPixels)) % destNumPixels;
			
		dest[index] = source[sourceRows * (img * destNumPixels + pix) + ch];
	}
} 
template<typename T>
void _cuda_rearrange(T *dest, const T *source, const unsigned int sourceRows,
		const unsigned int destRows, const unsigned int destColumns, const unsigned int destNumPixels)
{
	unsigned int nElements = destColumns * destRows;
	int gridSize = (int)ceil((float)nElements/ THREADS_PER_BLOCK);
	
	__cuda_rearrange<<<gridSize, THREADS_PER_BLOCK>>>(dest, source, sourceRows, destRows, destColumns, destNumPixels);
}
template __global__ void __cuda_rearrange(double *dest, const double *source, const unsigned int sourceRows,
		const unsigned int destRows, const unsigned int destColumns, const unsigned int destNumPixels);
template __global__ void __cuda_rearrange(float *dest, const float *source, const unsigned int sourceRows,
		const unsigned int destRows, const unsigned int destColumns, const unsigned int destNumPixels);
template void _cuda_rearrange(double *dest, const double *source, const unsigned int sourceRows,
		const unsigned int destRows, const unsigned int destColumns, const unsigned int destNumPixels);
template void _cuda_rearrange(float *dest, const float *source, const unsigned int sourceRows,
		const unsigned int destRows, const unsigned int destColumns, const unsigned int destNumPixels);
/*
 *
 * Rearrange back propogation
 *
 */
template<typename T> 
__global__ void __cuda_rearrangeBackProp(T *dest, const T *source, const unsigned int sourceColumns,
		const unsigned int destRows, const unsigned int destColumns, const unsigned int destNumPixels)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < (destRows * destColumns)) {
		unsigned int img = index / (destNumPixels * destRows);
		unsigned int pix = (index % (destNumPixels * destRows)) / destRows;
		unsigned int ch = (index % (destNumPixels * destRows)) % destRows;
		
		dest[index] = source[img*(destRows * destNumPixels) + ch * destNumPixels + pix];
	}
}
template<typename T>
void _cuda_rearrangeBackProp(T *dest, const T *source, const unsigned int sourceColumns,
		const unsigned int destRows, const unsigned int destColumns, const unsigned int destNumPixels)
{
	unsigned int nElements = destRows * destColumns;
	int gridSize = (int)ceil((float)nElements/THREADS_PER_BLOCK);
	
	__cuda_rearrangeBackProp<<<gridSize, THREADS_PER_BLOCK>>>(dest, source, sourceColumns, 
			destRows, destColumns, destNumPixels);
}		
template __global__ void __cuda_rearrangeBackProp(double *dest, const double *source, const unsigned int sourceColumns,
		const unsigned int destRows, const unsigned int destColumns, const unsigned int destNumPixels);
template __global__ void __cuda_rearrangeBackProp(float *dest, const float *source, const unsigned int sourceColumns,
		const unsigned int destRows, const unsigned int destColumns, const unsigned int destNumPixels);
template void _cuda_rearrangeBackProp(double *dest, const double *source, const unsigned int sourceColumns,
		const unsigned int destRows, const unsigned int destColumns, const unsigned int destNumPixels);
template void _cuda_rearrangeBackProp(float *dest, const float *source, const unsigned int sourceColumns,
		const unsigned int destRows, const unsigned int destColumns, const unsigned int destNumPixels);



/*
 *
 * argMax
 *
 *
 */
template<typename T>
__global__ void __cuda_argMax(T *matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int *resultDevPtr){
    unsigned  int column= threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumns){
	int beginCol = column * nRows;
	T maxVal = matrixPtr[beginCol];
	resultDevPtr[column] = 0;
	for (int i = 1; i < nRows; i++){
	    T val = matrixPtr[beginCol + i];
	    if (val > maxVal){
		maxVal =  val;
		resultDevPtr[column] = i;
	    }
	}
	}
}

template<typename T>
void _cuda_argMax(T *matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int *resultDevPtr)
{
    // parallelization over columns only
    int gridSize = (int)ceil( (float) nColumns/THREADS_PER_BLOCK);
    __cuda_argMax <<< gridSize, THREADS_PER_BLOCK>>> (matrixPtr, nRows, nColumns, resultDevPtr);
}

template __global__ void __cuda_argMax<double>(double *matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int *resultDevPtr);
template void _cuda_argMax<double>(double *matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int *resultDevPtr);
template __global__ void __cuda_argMax<float>(float *matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int *resultDevPtr);
template void _cuda_argMax<float>(float *matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int *resultDevPtr);


/*
 *
 * max
 * set max per column to 1.0, all other to 0.0
 *
 */
template<typename T>
__global__ void __cuda_max(T *devResult, unsigned int nRows, unsigned int nColumns){
    unsigned  int column = threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumns) {
        unsigned int argMax = 0;
        T max = devResult[column * nRows];
        for (int i = 0; i < nRows; i++) {
            if (devResult[column * nRows + i] > max) {
                max = devResult[column * nRows + i];
                argMax = i;
            }
            devResult[column * nRows + i] = 0.0;
        }
        devResult[column * nRows + argMax] = 1.0;
    }
}

template<typename T>
void _cuda_max(T *devResult, unsigned int nRows, unsigned int nColumns)
{
    int gridSize = (int)ceil( (float) nColumns/THREADS_PER_BLOCK);
    __cuda_max <<< gridSize, THREADS_PER_BLOCK>>> (devResult, nRows, nColumns);
}

template __global__ void __cuda_max<double>(double *devResult, unsigned int nRows, unsigned int nColumns);
template void _cuda_max<double>(double *devResult, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_max<float>(float *devResult, unsigned int nRows, unsigned int nColumns);
template void _cuda_max<float>(float *devResult, unsigned int nRows, unsigned int nColumns);

/*
 *
 * max
 *
 *
 */
template<typename T>
__global__ void __cuda_max(T *devResult, const T *devA, const T *devB, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements) {
        if (devA[index] < devB[index])
            devResult[index] = devB[index];
        else
            devResult[index] = devA[index];
    }
}

template<typename T>
void _cuda_max(T *devResult, const T *devA, const T *devB, unsigned int nRows, unsigned int nColumns)
{
    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);
    __cuda_max <<< gridSize, THREADS_PER_BLOCK>>> (devResult, devA, devB, nElements);
}

template __global__ void __cuda_max<double>(double *devResult, const double *devA, const double *devB, unsigned int nElements);
template void _cuda_max<double>(double *devResult, const double *devA, const double *devB, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_max<float>(float *devResult, const float *devA, const float *devB, unsigned int nElements);
template void _cuda_max<float>(float *devResult, const float *devA, const float *devB, unsigned int nRows, unsigned int nColumns);

/*
 *
 * elementwiseMultiplicationWithKroneckerDelta
 *
 *
 */
template<typename T>
__global__ void __cuda_elementwiseMultiplicationWithKroneckerDelta(T *devResult, const T *devA, const T *devB, unsigned int nElements){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements) {
        if (devA[index] != devB[index])
            devResult[index] = 0;
    }
}

template<typename T>
void _cuda_elementwiseMultiplicationWithKroneckerDelta(T *devResult, const T *devA, const T *devB, unsigned int nRows, unsigned int nColumns)
{
    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);
    __cuda_elementwiseMultiplicationWithKroneckerDelta <<< gridSize, THREADS_PER_BLOCK>>> (devResult, devA, devB, nElements);
}

template __global__ void __cuda_elementwiseMultiplicationWithKroneckerDelta<double>(double *devResult, const double *devA, const double *devB, unsigned int nElements);
template void _cuda_elementwiseMultiplicationWithKroneckerDelta<double>(double *devResult, const double *devA, const double *devB, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_elementwiseMultiplicationWithKroneckerDelta<float>(float *devResult, const float *devA, const float *devB, unsigned int nElements);
template void _cuda_elementwiseMultiplicationWithKroneckerDelta<float>(float *devResult, const float *devA, const float *devB, unsigned int nRows, unsigned int nColumns);

/*
 *
 * nClassificationErrors
 *
 *
 */
template<typename T>
__global__ void __cuda_nClassificationErrors(T *matrixPtr, unsigned int nRows, unsigned int nColumns, T *targets, unsigned int *resultDevPtr){
    unsigned  int column= threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumns){
	int beginCol = column * nRows;
	T maxVal = matrixPtr[beginCol];
	uint argmax = 0;
	for (int i = 1; i < nRows; i++){
	    T val = matrixPtr[beginCol + i];
	    if (val > maxVal){
		maxVal =  val;
		argmax = i;
	    }
	}
	if (targets[nRows * column + argmax] != 1.0){
	    atomicAdd(resultDevPtr, 1);
	}
    }
}

template<typename T>
void _cuda_nClassificationErrors(T *matrixPtr, unsigned int nRows, unsigned int nColumns, T *targets, unsigned int *resultDevPtr)
{
    // parallelization over columns only
    int gridSize = (int)ceil( (float) nColumns/THREADS_PER_BLOCK);
    unsigned int result = 0;
    hipMemcpy(resultDevPtr, &result, sizeof(unsigned int), hipMemcpyHostToDevice);
    __cuda_nClassificationErrors <<< gridSize, THREADS_PER_BLOCK>>> (matrixPtr, nRows, nColumns, targets, resultDevPtr);
}


template __global__ void __cuda_nClassificationErrors<double>(double *matrixPtr, unsigned int nRows, unsigned int nColumns, double *targets, unsigned int *resultDevPtr);
template void _cuda_nClassificationErrors<double>(double *matrixPtr, unsigned int nRows, unsigned int nColumns, double *targets, unsigned int *resultDevPtr);
template __global__ void __cuda_nClassificationErrors<float>(float *matrixPtr, unsigned int nRows, unsigned int nColumns, float *targets, unsigned int *resultDevPtr);
template void _cuda_nClassificationErrors<float>(float *matrixPtr, unsigned int nRows, unsigned int nColumns, float *targets, unsigned int *resultDevPtr);

// crossEntropyObjectiveFunction
template<typename T>
__global__ void __cuda_crossEntropyObjectiveFunction(T *matrixPtr, unsigned int nRows, unsigned int nColumns, T *targets, T *objFctn){
    unsigned  int column= threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumns){
    objFctn[column] = 0;
    for (int i = 0; i < nRows; i++){
        if (targets[nRows * column + i] == 1.0)
            objFctn[column] -= log(matrixPtr[nRows * column + i]);
    }
    }
}

template<typename T>
void _cuda_crossEntropyObjectiveFunction(T *matrixPtr, unsigned int nRows, unsigned int nColumns, T* targets, T *objFctn)
{   
    // parallelization over columns only
    int gridSize = (int)ceil( (float) nColumns/THREADS_PER_BLOCK);
    __cuda_crossEntropyObjectiveFunction <<< gridSize , THREADS_PER_BLOCK >>> (matrixPtr, nRows, nColumns, targets, objFctn);
}

template __global__ void __cuda_crossEntropyObjectiveFunction<double>(double *matrixPtr, unsigned int nRows, unsigned int nColumns, double *targets, double *objFctn);
template void _cuda_crossEntropyObjectiveFunction<double>(double *matrixPtr, unsigned int nRows, unsigned int nColumns, double *targets, double *objFctn);
template __global__ void __cuda_crossEntropyObjectiveFunction<float>(float *matrixPtr, unsigned int nRows, unsigned int nColumns, float *targets, float *objFctn);
template void _cuda_crossEntropyObjectiveFunction<float>(float *matrixPtr, unsigned int nRows, unsigned int nColumns, float *targets, float *objFctn);

// weightedCrossEntropyObjectiveFunction
template<typename T>
__global__ void __cuda_weightedCrossEntropyObjectiveFunction(T *matrixPtr, unsigned int nRows, unsigned int nColumns, T *targets, T *objFctn, T *weights){
    unsigned  int column= threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumns){
    objFctn[column] = 0;
    for (int i = 0; i < nRows; i++){
        if (targets[nRows * column + i] == 1.0)
            objFctn[column] -= log(matrixPtr[nRows * column + i]) * weights[column];
    }
    }
}

template<typename T>
void _cuda_weightedCrossEntropyObjectiveFunction(T *matrixPtr, unsigned int nRows, unsigned int nColumns, T *targets, T *objFctn, T *weights)
{
    // parallelization over columns only
    int gridSize = (int)ceil( (float) nColumns/THREADS_PER_BLOCK);
    __cuda_weightedCrossEntropyObjectiveFunction <<< gridSize , THREADS_PER_BLOCK >>> (matrixPtr, nRows, nColumns, targets, objFctn, weights);
}

template __global__ void __cuda_weightedCrossEntropyObjectiveFunction<double>(double *matrixPtr, unsigned int nRows, unsigned int nColumns, double *targets, double *objFctn, double *weights);
template void _cuda_weightedCrossEntropyObjectiveFunction<double>(double *matrixPtr, unsigned int nRows, unsigned int nColumns, double *targets, double *objFctn, double *weights);
template __global__ void __cuda_weightedCrossEntropyObjectiveFunction<float>(float *matrixPtr, unsigned int nRows, unsigned int nColumns, float *targets, float *objFctn, float *weights);
template void _cuda_weightedCrossEntropyObjectiveFunction<float>(float *matrixPtr, unsigned int nRows, unsigned int nColumns, float *targets, float *objFctn, float *weights);



// squaredErrorObjectiveFunction

template<typename T>
__global__ void __cuda_squaredErrorObjectiveFunction(T *matrixPtr, unsigned int nRows, unsigned int nColumns, T *targets, T *objFctn){
    unsigned int column = threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumns){
	objFctn[column] = 0.0f;
	for (int row = 0; row < nRows; row++){
	    unsigned int position = column * nRows + row;
	    objFctn[column] += (matrixPtr[position] - targets[position]) * (matrixPtr[position] - targets[position]);
	}
    }
}

template<typename T>
void _cuda_squaredErrorObjectiveFunction(T *matrixPtr, unsigned int nRows, unsigned int nColumns, T *targets, T *objFctn)
{
    int gridSize = (int)ceil( (float) nColumns/THREADS_PER_BLOCK);
    __cuda_squaredErrorObjectiveFunction <<< gridSize , THREADS_PER_BLOCK >>> (matrixPtr, nRows, nColumns, targets, objFctn);
}


template __global__ void __cuda_squaredErrorObjectiveFunction(double *matrixPtr, unsigned int nRows, unsigned int nColumns, double *targets, double *objFctn);
template void _cuda_squaredErrorObjectiveFunction(double *matrixPtr, unsigned int nRows, unsigned int nColumns, double *targets, double *objFctn);
template __global__ void __cuda_squaredErrorObjectiveFunction(float *matrixPtr, unsigned int nRows, unsigned int nColumns, float *targets, float *objFctn);
template void _cuda_squaredErrorObjectiveFunction(float *matrixPtr, unsigned int nRows, unsigned int nColumns, float *targets, float *objFctn);

// weightedSquaredErrorObjectiveFunction

template<typename T>
__global__ void __cuda_weightedSquaredErrorObjectiveFunction(T *matrixPtr, unsigned int nRows, unsigned int nColumns, T *targets, T *objFctn, T *weights){
    unsigned int column = threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumns){
	objFctn[column] = 0.0f;
	for (int row = 0; row < nRows; row++){
	    unsigned int position = column * nRows + row;
	    objFctn[column] += (matrixPtr[position] - targets[position]) * (matrixPtr[position] - targets[position]);
	}
	objFctn[column] *= weights[column];
    }
}

template<typename T>
void _cuda_weightedSquaredErrorObjectiveFunction(T *matrixPtr, unsigned int nRows, unsigned int nColumns, T *targets, T *objFctn, T *weights)
{
    int gridSize = (int)ceil( (float) nColumns/THREADS_PER_BLOCK);
    __cuda_weightedSquaredErrorObjectiveFunction <<< gridSize , THREADS_PER_BLOCK >>> (matrixPtr, nRows, nColumns, targets, objFctn, weights);
}

template __global__ void __cuda_weightedSquaredErrorObjectiveFunction(double *matrixPtr, unsigned int nRows, unsigned int nColumns, double *targets, double *objFctn, double *weights);
template void _cuda_weightedSquaredErrorObjectiveFunction(double *matrixPtr, unsigned int nRows, unsigned int nColumns, double *targets, double *objFctn, double *weights);
template __global__ void __cuda_weightedSquaredErrorObjectiveFunction(float *matrixPtr, unsigned int nRows, unsigned int nColumns, float *targets, float *objFctn, float *weights);
template void _cuda_weightedSquaredErrorObjectiveFunction(float *matrixPtr, unsigned int nRows, unsigned int nColumns, float *targets, float *objFctn, float *weights);


// smoothedL1ObjectiveFunction

template<typename T>
__global__ void __cuda_smoothedL1ObjectiveFunction(T *matrixPtr, unsigned int nRows, unsigned int nColumns, T *targets, T *objFctn){
    unsigned int column = threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumns){
	objFctn[column] = 0.0f;
	for (int row = 0; row < nRows; row++){
	    unsigned int position = column * nRows + row;
	    if ((matrixPtr[position] - targets[position]) < -1.0)
	    	objFctn[column] += (targets[position] - matrixPtr[position]) - 0.5;
	    else if ((matrixPtr[position] - targets[position]) > 1.0)
	        objFctn[column] += (matrixPtr[position] - targets[position]) - 0.5;
	    else
	        objFctn[column] += 0.5 * (matrixPtr[position] - targets[position]) * (matrixPtr[position] - targets[position]);
	}
    }
}

template<typename T>
void _cuda_smoothedL1ObjectiveFunction(T *matrixPtr, unsigned int nRows, unsigned int nColumns, T *targets, T *objFctn)
{
    int gridSize = (int)ceil( (float) nColumns/THREADS_PER_BLOCK);
    __cuda_smoothedL1ObjectiveFunction <<< gridSize , THREADS_PER_BLOCK >>> (matrixPtr, nRows, nColumns, targets, objFctn);
}


template __global__ void __cuda_smoothedL1ObjectiveFunction(double *matrixPtr, unsigned int nRows, unsigned int nColumns, double *targets, double *objFctn);
template void _cuda_smoothedL1ObjectiveFunction(double *matrixPtr, unsigned int nRows, unsigned int nColumns, double *targets, double *objFctn);
template __global__ void __cuda_smoothedL1ObjectiveFunction(float *matrixPtr, unsigned int nRows, unsigned int nColumns, float *targets, float *objFctn);
template void _cuda_smoothedL1ObjectiveFunction(float *matrixPtr, unsigned int nRows, unsigned int nColumns, float *targets, float *objFctn);


// weightedSmoothedL1ObjectiveFunction

template<typename T>
__global__ void __cuda_weightedSmoothedL1ObjectiveFunction(T *matrixPtr, unsigned int nRows, unsigned int nColumns, T *targets, T* weights, T *objFctn){
    unsigned int column = threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumns){
	objFctn[column] = 0.0f;
	for (int row = 0; row < nRows; row++){
	    unsigned int position = column * nRows + row;
	    if ((matrixPtr[position] - targets[position]) < -1.0)
	    	objFctn[column] += (targets[position] - matrixPtr[position]) - 0.5;
	    else if ((matrixPtr[position] - targets[position]) > 1.0)
	        objFctn[column] += (matrixPtr[position] - targets[position]) - 0.5;
	    else
	        objFctn[column] += 0.5 * (matrixPtr[position] - targets[position]) * (matrixPtr[position] - targets[position]);
	}
	objFctn[column] *= weights[column];
    }
}

template<typename T>
void _cuda_weightedSmoothedL1ObjectiveFunction(T *matrixPtr, unsigned int nRows, unsigned int nColumns, T *targets, T* weights, T *objFctn)
{
    int gridSize = (int)ceil( (float) nColumns/THREADS_PER_BLOCK);
    __cuda_weightedSmoothedL1ObjectiveFunction <<< gridSize , THREADS_PER_BLOCK >>> (matrixPtr, nRows, nColumns, targets, weights, objFctn);
}


template __global__ void __cuda_weightedSmoothedL1ObjectiveFunction(double *matrixPtr, unsigned int nRows, unsigned int nColumns, double *targets, double *objFctn, double *weights);
template void _cuda_weightedSmoothedL1ObjectiveFunction(double *matrixPtr, unsigned int nRows, unsigned int nColumns, double *targets, double *objFctn, double *weights);
template __global__ void __cuda_weightedSmoothedL1ObjectiveFunction(float *matrixPtr, unsigned int nRows, unsigned int nColumns, float *targets, float *objFctn, float *weights);
template void _cuda_weightedSmoothedL1ObjectiveFunction(float *matrixPtr, unsigned int nRows, unsigned int nColumns, float *targets, float *objFctn, float *weights);


/*
 *  appendSecondOrderFeatures
 */

template<typename T>
__global__ void __cuda_appendSecondOrderFeatures(const T *X, unsigned int nRowsX, unsigned int nColumnsX, T *Y, unsigned int nRowsY, unsigned int offset){
    unsigned  int column = threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumnsX){
	unsigned int pos = offset;
	for (unsigned int i = 0; i < nRowsX; ++ i) {
	    for (unsigned int j = i; j < nRowsX; ++ j) {
		Y[column * nRowsY + pos] = X[column * nRowsX + i] * X[column * nRowsX + j];
		pos++;
	    }
	}
    }
}

template<typename T>
void _cuda_appendSecondOrderFeatures(const T *X, unsigned int nRowsX, unsigned int nColumnsX, T *Y, unsigned int nRowsY, unsigned int offset){
    int gridSize = (int)ceil( (float) nColumnsX/THREADS_PER_BLOCK);
    __cuda_appendSecondOrderFeatures <<< gridSize , THREADS_PER_BLOCK >>> (X, nRowsX, nColumnsX, Y, nRowsY, offset);
}

template __global__ void __cuda_appendSecondOrderFeatures(const double *X, unsigned int nRowsX, unsigned int nColumnsX, double *Y, unsigned int nRowsY, unsigned int offset);
template void _cuda_appendSecondOrderFeatures(const double *X, unsigned int nRowsX, unsigned int nColumnsX, double *Y, unsigned int nRowsY, unsigned int offset);
template __global__ void __cuda_appendSecondOrderFeatures(const float *X, unsigned int nRowsX, unsigned int nColumnsX, float *Y, unsigned int nRowsY, unsigned int offset);
template void _cuda_appendSecondOrderFeatures(const float *X, unsigned int nRowsX, unsigned int nColumnsX, float *Y, unsigned int nRowsY, unsigned int offset);

/*
 *  appendDiagonalSecondOrderFeatures
 */

template<typename T>
__global__ void __cuda_appendDiagonalSecondOrderFeatures(const T *X, unsigned int nRowsX, unsigned int nColumnsX, T *Y, unsigned int nRowsY, unsigned int offset){
    unsigned  int column = threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumnsX){
	unsigned int pos = offset;
	for (unsigned int i = 0; i < nRowsX; ++ i) {
		Y[column * nRowsY + pos] = X[column * nRowsX + i] * X[column * nRowsX + i];
		pos++;
	}
    }
}

template<typename T>
void _cuda_appendDiagonalSecondOrderFeatures(const T *X, unsigned int nRowsX, unsigned int nColumnsX, T *Y, unsigned int nRowsY, unsigned int offset){
    int gridSize = (int)ceil( (float) nColumnsX/THREADS_PER_BLOCK);
    __cuda_appendDiagonalSecondOrderFeatures <<< gridSize , THREADS_PER_BLOCK >>> (X, nRowsX, nColumnsX, Y, nRowsY, offset);
}

template __global__ void __cuda_appendDiagonalSecondOrderFeatures(const double *X, unsigned int nRowsX, unsigned int nColumnsX, double *Y, unsigned int nRowsY, unsigned int offset);
template void _cuda_appendDiagonalSecondOrderFeatures(const double *X, unsigned int nRowsX, unsigned int nColumnsX, double *Y, unsigned int nRowsY, unsigned int offset);
template __global__ void __cuda_appendDiagonalSecondOrderFeatures(const float *X, unsigned int nRowsX, unsigned int nColumnsX, float *Y, unsigned int nRowsY, unsigned int offset);
template void _cuda_appendDiagonalSecondOrderFeatures(const float *X, unsigned int nRowsX, unsigned int nColumnsX, float *Y, unsigned int nRowsY, unsigned int offset);


// appendThirdOrderFeatures

template<typename T>
__global__ void __cuda_appendThirdOrderFeatures(const T *X, unsigned int nRowsX, unsigned int nColumnsX, T *Y, unsigned int nRowsY, unsigned int offset){
    unsigned  int column = threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumnsX){
	unsigned int pos = offset;
	for (unsigned int i = 0; i < nRowsX; ++ i) {
	    for (unsigned int j = i; j < nRowsX; ++ j) {
		for (unsigned int k = j; k < nRowsX; ++ k) {
		    Y[column * nRowsY + pos]  = X[column * nRowsX + i] * X[column * nRowsX + j] * X[column * nRowsX + k];
		    pos++;
		}
	    }
	}
    }
}

template<typename T>
void _cuda_appendThirdOrderFeatures(const T *X, unsigned int nRowsX, unsigned int nColumnsX, T *Y, unsigned int nRowsY, unsigned int offset){
    int gridSize = (int)ceil( (float) nColumnsX/THREADS_PER_BLOCK);
    __cuda_appendThirdOrderFeatures <<< gridSize , THREADS_PER_BLOCK >>> (X, nRowsX, nColumnsX, Y, nRowsY, offset);
}

template __global__ void __cuda_appendThirdOrderFeatures(const double *X, unsigned int nRowsX, unsigned int nColumnsX, double *Y, unsigned int nRowsY, unsigned int offset);
template void _cuda_appendThirdOrderFeatures(const double *X, unsigned int nRowsX, unsigned int nColumnsX, double *Y, unsigned int nRowsY, unsigned int offset);
template __global__ void __cuda_appendThirdOrderFeatures(const float *X, unsigned int nRowsX, unsigned int nColumnsX, float *Y, unsigned int nRowsY, unsigned int offset);
template void _cuda_appendThirdOrderFeatures(const float *X, unsigned int nRowsX, unsigned int nColumnsX, float *Y, unsigned int nRowsY, unsigned int offset);

// appendDiagonalThirdOrderFeatures

template<typename T>
__global__ void __cuda_appendDiagonalThirdOrderFeatures(const T *X, unsigned int nRowsX, unsigned int nColumnsX, T *Y, unsigned int nRowsY, unsigned int offset){
    unsigned  int column = threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumnsX){
	unsigned int pos = offset;
	for (unsigned int i = 0; i < nRowsX; ++ i) {
		Y[column * nRowsY + pos]  = X[column * nRowsX + i] * X[column * nRowsX + i] * X[column * nRowsX + i];
		pos++;
	}
    }
}

template<typename T>
void _cuda_appendDiagonalThirdOrderFeatures(const T *X, unsigned int nRowsX, unsigned int nColumnsX, T *Y, unsigned int nRowsY, unsigned int offset){
    int gridSize = (int)ceil( (float) nColumnsX/THREADS_PER_BLOCK);
    __cuda_appendDiagonalThirdOrderFeatures <<< gridSize , THREADS_PER_BLOCK >>> (X, nRowsX, nColumnsX, Y, nRowsY, offset);
}

template __global__ void __cuda_appendDiagonalThirdOrderFeatures(const double *X, unsigned int nRowsX, unsigned int nColumnsX, double *Y, unsigned int nRowsY, unsigned int offset);
template void _cuda_appendDiagonalThirdOrderFeatures(const double *X, unsigned int nRowsX, unsigned int nColumnsX, double *Y, unsigned int nRowsY, unsigned int offset);
template __global__ void __cuda_appendDiagonalThirdOrderFeatures(const float *X, unsigned int nRowsX, unsigned int nColumnsX, float *Y, unsigned int nRowsY, unsigned int offset);
template void _cuda_appendDiagonalThirdOrderFeatures(const float *X, unsigned int nRowsX, unsigned int nColumnsX, float *Y, unsigned int nRowsY, unsigned int offset);

/*
 *
 * gaussianMixturePosteriors
 * computes unnormalized, unexponentiated Gaussian mixture posteriors
 * -> p(c|x) can be obtained with application of softmax on the result of this function
 *
 */
template<typename T>
__global__ void __cuda_gaussianMixturePosteriors(T *P, const T *X, const T *means, const T *variances, const T *weights, unsigned int nFeatures, unsigned int featureDim, unsigned int nMixtures){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nFeatures * nMixtures) {
        unsigned int k = index % nMixtures;
        unsigned int n = index / nMixtures;
	    T expn = 0;
	    T det = 0;
	    for (unsigned int d = 0; d < featureDim; d++) {
	        expn += (X[n * featureDim + d] - means[d * nMixtures + k]) * (X[n * featureDim + d] - means[d * nMixtures + k])
	                / variances[d * nMixtures + k];
	        det += log(variances[d * nMixtures + k]);
	    }
	    P[index] = log(weights[k]) - 0.5 * expn - 0.5 * log(2 * HIP_PI) * featureDim - 0.5 * det;
	}
}

template<typename T>
void _cuda_gaussianMixturePosteriors(T *P, const T *X, const T *means, const T *variances, const T *weights, unsigned int nFeatures, unsigned int featureDim, unsigned int nMixtures)
{

    unsigned int nElements = nFeatures * nMixtures;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_gaussianMixturePosteriors <<< gridSize , THREADS_PER_BLOCK >>> (P, X, means, variances, weights, nFeatures, featureDim, nMixtures);
}

template __global__ void __cuda_gaussianMixturePosteriors(double *P, const double *X, const double *means, const double *variances, const double *weights, unsigned int nFeatures, unsigned int featureDim, unsigned int nMixtures);
template void _cuda_gaussianMixturePosteriors(double *P, const double *X, const double *means, const double *variances, const double *weights, unsigned int nFeatures, unsigned int featureDim, unsigned int nMixtures);
template __global__ void __cuda_gaussianMixturePosteriors(float *P, const float *X, const float *means, const float *variances, const float *weights, unsigned int nFeatures, unsigned int featureDim, unsigned int nMixtures);
template void _cuda_gaussianMixturePosteriors(float *P, const float *X, const float *means, const float *variances, const float *weights, unsigned int nFeatures, unsigned int featureDim, unsigned int nMixtures);

/*
 *
 * fisher encoding
 *
 */
template<typename T>
__global__ void __cuda_fisherEncoding(T *F, const T *X, const T *means, const T *variances, const T *weights, const T* gamma, unsigned int nFeatures, unsigned int featureDim, unsigned int nMixtures){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nFeatures * nMixtures * featureDim) {
        unsigned int n = index / (nMixtures * featureDim);
        unsigned int k = (index % (nMixtures * featureDim)) / featureDim;
        unsigned int d = (index % (nMixtures * featureDim)) % featureDim;
        // first order component
        F[d + k * featureDim + n * featureDim * nMixtures * 2] = gamma[k + n * nMixtures]
                * (X[d + n * featureDim] - means[k + d * nMixtures]) / sqrt(variances[k + d * nMixtures] * weights[k]);
        // second order component
        F[d + (k + nMixtures) * featureDim + n * featureDim * nMixtures * 2] = gamma[k + n * nMixtures]
                * ( (X[d + n * featureDim] - means[k + d * nMixtures]) * (X[d + n * featureDim] - means[k + d * nMixtures])
                     / variances[k + d * nMixtures] - 1.0 )
                / sqrt(2 * weights[k]);
	}
}

template<typename T>
void _cuda_fisherEncoding(T *F, const T *X, const T *means, const T *variances, const T *weights, const T *gamma, unsigned int nFeatures, unsigned int featureDim, unsigned int nMixtures)
{

    unsigned int nElements = nFeatures * nMixtures * featureDim;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_fisherEncoding <<< gridSize , THREADS_PER_BLOCK >>> (F, X, means, variances, weights, gamma, nFeatures, featureDim, nMixtures);
}

template __global__ void __cuda_fisherEncoding(double *F, const double *X, const double *means, const double *variances, const double *weights, const double *gamma, unsigned int nFeatures, unsigned int featureDim, unsigned int nMixtures);
template void _cuda_fisherEncoding(double *F, const double *X, const double *means, const double *variances, const double *weights, const double *gamma, unsigned int nFeatures, unsigned int featureDim, unsigned int nMixtures);
template __global__ void __cuda_fisherEncoding(float *F, const float *X, const float *means, const float *variances, const float *weights, const float *gamma, unsigned int nFeatures, unsigned int featureDim, unsigned int nMixtures);
template void _cuda_fisherEncoding(float *F, const float *X, const float *means, const float *variances, const float *weights, const float *gamma, unsigned int nFeatures, unsigned int featureDim, unsigned int nMixtures);

/*
 *
 * dropout
 *
 */
template<typename T>
__global__ void __cuda_dropout(T *data, const T *mask, unsigned int nElements, T dropoutProbability){
    unsigned  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if ((index < nElements) && (mask[index] < dropoutProbability))
	    data[index] = 0.0;
}

template<typename T>
void _cuda_dropout(T *data, const T *mask, unsigned int nRows, unsigned int nColumns, T dropoutProbability)
{

    unsigned int nElements = nRows * nColumns;
    int gridSize = (int)ceil( (float) nElements/THREADS_PER_BLOCK);

    __cuda_dropout <<< gridSize , THREADS_PER_BLOCK >>> (data, mask, nElements, dropoutProbability);
}

template __global__ void __cuda_dropout(double *data, const double *mask, unsigned int nElements, double dropoutProbability);
template void _cuda_dropout(double *data, const double *mask, unsigned int nRows, unsigned int nColumns, double dropoutProbability);
template __global__ void __cuda_dropout(float *data, const float *mask, unsigned int nElements, float dropoutProbability);
template void _cuda_dropout(float *data, const float *mask, unsigned int nRows, unsigned int nColumns, float dropoutProbability);
